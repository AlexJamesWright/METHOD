#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "srmhd.h"
#include "simulation.h"
#include "simData.h"
#include "initFunc.h"
#include <cstdlib>
#include <cmath>

namespace {

  /* ##################### Test default model constructor ####################*/

  TEST(SRMHD, DefaultConstructor)
  {
    SRMHD model;
    EXPECT_EQ(model.Ncons, 9);
    EXPECT_EQ(model.Nprims, 8);
    EXPECT_EQ(model.Naux, 10);
  }




  /* ######################### Test model constructor ########################*/

  TEST(SRMHD, Constructor)
  {
    Data d(100, 10, 0, 1, -0.5, 0.5, 0.8);
    SRMHD model(&d);
    EXPECT_EQ(model.data->Ncons, 9);
    EXPECT_EQ(model.data->Nprims, 8);
    EXPECT_EQ(model.data->Naux, 10);

  }



  /* ######################### Test prims2all transform ########################*/

  TEST(SRMHD, PrimsToAll)
  {
    // Set up
    Data d(10, 10, 0, 1, 0, 1, 1.0);
    SRMHD model(&d);
    Simulation sim(&d);
    InitialFunc init(&d);

    // Set cons and aux vars to rubbish so we know they have changed, and
    // ser all primitive variables to (nearly) zero: 0 will give zero-division error
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int var(0); var < d.Nprims; var++) {
          d.prims[d.id(var, i, j)] = 1e-16;
        }
        for (int var(0); var < d.Ncons; var++) {
          d.cons[d.id(var, i, j)] = 3.1415926;
        }
        for (int var(0); var < d.Naux; var++) {
          d.aux[d.id(var, i, j)] = 2.718281828;
        }
      }
    }

    // Quick check that worked
    EXPECT_EQ(d.cons[d.id(6, 2, 5)], 3.1415926);
    EXPECT_EQ(d.aux[d.id(7, 2, 5)], 2.718281828);
    EXPECT_EQ(d.prims[d.id(7, 8, 4)], 1e-16);

    // Apply conversion and check all cons and aux are zero except h and W
    model.primsToAll(d.cons, d.prims, d.aux);

    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        // Conserved
        for (int var(0); var < d.Ncons; var++) {
          EXPECT_NEAR(d.cons[d.id(var, i, j)], 0.0, 1e-15);
        }
        // Auxilliary
        for (int var(0); var < d.Naux; var++) {
          if (var == 0) EXPECT_EQ(d.aux[d.id(var, i, j)], 3.5);
          else if (var == 1) EXPECT_NEAR(d.aux[d.id(var, i, j)], 1.0, 1e-15);
          else if (var == 2) EXPECT_EQ(d.aux[d.id(var, i, j)], 1.5);
          else if (var == 3) EXPECT_EQ(d.aux[d.id(var, i, j)], sqrt(10.0/21.0));
          else EXPECT_NEAR(d.aux[d.id(var, i, j)], 0.0, 1e-15);
        }
      }
    }
  }


  /* ######################### Test source contribution ########################*/

  TEST(SRMHD, SourceTerm)
  {

    // Set up
    Data d(10, 10, 0, 1, 0, 1, 1.0, 0.5, 4, 5.0/3.0, 0.0, 0, 0, 0, 0, 0.5);
    SRMHD model(&d);
    Simulation sim(&d);

    // Set cons to something
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int var(0); var < d.Ncons; var++) {
          d.cons[d.id(var, i, j)] = 3.1415926;
        }
      }
    }
    // Determine source
    model.sourceTerm(d.cons, d.prims, d.aux, d.source);
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int var(0); var < d.Ncons; var++) {
          if (var == 8) EXPECT_EQ(d.source[d.id(var, i, j)], -3.1415926 / (0.25));
          else EXPECT_EQ(d.source[d.id(var, i, j)], 0);
        }
      }
    }
  }


  /* ######################### Test flux vector splitting ########################*/

  TEST(SRMHD, FluxVectorSplittingStationary)
  {

    // Set up
    Data d(10, 10, 0, 1, 0, 1, 1.0, 0.5, 4, 5.0/3.0, 0.0, 0, 0, 0, 0, 0.5);
    SRMHD model(&d);
    Simulation sim(&d);

    // Set state to stationary equilibrium state
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        d.prims[d.id(0, i, j)] = 0.5; // Require non-zero density
        d.prims[d.id(1, i, j)] = 0.0;
        d.prims[d.id(2, i, j)] = 0.0;
        d.prims[d.id(3, i, j)] = 0.0;
        d.prims[d.id(4, i, j)] = 0.0;
        d.prims[d.id(5, i, j)] = 0.0;
        d.prims[d.id(6, i, j)] = 0.0;
        d.prims[d.id(7, i, j)] = 0.0;
        d.prims[d.id(8, i, j)] = 0.0;
      }
    }

    model.primsToAll(d.cons, d.prims, d.aux);

    // System is stationary, there should be zero flux
    // x-direction
    model.fluxFunc(d.cons, d.prims, d.aux, d.f, d.fnet, 0);
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int var(0); var < d.Ncons; var++) {
          EXPECT_EQ(d.fnet[d.id(var, i, j)], 0.0);
        }
      }
    }
    // y-direction
    model.fluxFunc(d.cons, d.prims, d.aux, d.f, d.fnet, 1);
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int var(0); var < d.Ncons; var++) {
          EXPECT_EQ(d.fnet[d.id(var, i, j)], 0.0);
        }
      }
    }
  }


}
