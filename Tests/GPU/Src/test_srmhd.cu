#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "srmhd.h"
#include "simulation.h"
#include "simData.h"
#include "initFunc.h"
#include "fluxVectorSplitting.h"
#include <cstdlib>
#include <cmath>
#include <stdio.h>

/* ######################### Test model constructor ########################*/

TEST(SRMHD, Constructor)
{
  Data d(100, 10, 0, 0, 1, -0.5, 0.5, -0.1, 0.1, 0.8);
  SRMHD model(&d);
  EXPECT_EQ(d.Ncons, 9);
  EXPECT_EQ(d.Nprims, 8);
  EXPECT_EQ(d.Naux, 13);

}



/* ######################### Test flux vector splitting ########################*/

TEST(SRMHD, FluxVectorSplittingStationary)
{

  // Set up
  Data d(10, 10, 10, 0, 1, 0, 1, 0, 1, 1.0, 0.5, 4, 5.0/3.0, 1000.0, 0.5);
  SRMHD model(&d);
  FVS fluxMethod(&d, &model);
  Simulation sim(&d);

  // Set state to stationary equilibrium state
  for (int i(0); i < d.Nx; i++) {
    for (int j(0); j < d.Ny; j++) {
      for (int k(0); k < d.Nz; k++) {
        d.prims[d.id(0, i, j, k)] = 0.5; // Require non-zero density
        d.prims[d.id(1, i, j, k)] = 0.0;
        d.prims[d.id(2, i, j, k)] = 0.0;
        d.prims[d.id(3, i, j, k)] = 0.0;
        d.prims[d.id(4, i, j, k)] = 0.0;
        d.prims[d.id(5, i, j, k)] = 0.0;
        d.prims[d.id(6, i, j, k)] = 0.0;
        d.prims[d.id(7, i, j, k)] = 0.0;
        d.prims[d.id(8, i, j, k)] = 0.0;
      }
    }
  }

  model.primsToAll(d.cons, d.prims, d.aux);

  // System is stationary, there should be zero flux
  // x-direction
  fluxMethod.fluxReconstruction(d.cons, d.prims, d.aux, d.f, d.fnet, 0);
  for (int i(d.Ng); i < d.Nx-d.Ng; i++) {
    for (int j(d.Ng); j < d.Ny-d.Ng; j++) {
      for (int k(d.Ng); k < d.Nz-d.Ng; k++) {
        for (int var(0); var < d.Ncons; var++) {
          EXPECT_EQ(d.fnet[d.id(var, i, j, k)], 0.0);
        }
      }
    }
  }
  // y-direction
  fluxMethod.fluxReconstruction(d.cons, d.prims, d.aux, d.f, d.fnet, 0);
  for (int i(d.Ng); i < d.Nx-d.Ng; i++) {
    for (int j(d.Ng); j < d.Ny-d.Ng; j++) {
      for (int k(d.Ng); k < d.Nz-d.Ng; k++) {
        for (int var(0); var < d.Ncons; var++) {
          EXPECT_EQ(d.fnet[d.id(var, i, j, k)], 0.0);
        }
      }
    }
  }
  // z-direction
  fluxMethod.fluxReconstruction(d.cons, d.prims, d.aux, d.f, d.fnet, 0);
  for (int i(d.Ng); i < d.Nx-d.Ng; i++) {
    for (int j(d.Ng); j < d.Ny-d.Ng; j++) {
      for (int k(d.Ng); k < d.Nz-d.Ng; k++) {
        for (int var(0); var < d.Ncons; var++) {
          EXPECT_EQ(d.fnet[d.id(var, i, j, k)], 0.0);
        }
      }
    }
  }
}


/* ######################### Test source contribution ########################*/

TEST(SRMHD, SourceTerm)
{

  // Set up
  Data d(10, 10, 10, 0, 1, 0, 1, 0, 1, 1.0, 0.5, 4, 5.0/3.0, 1000.0, 0.5);
  SRMHD model(&d);
  Simulation sim(&d);

  // Set cons to something
  for (int i(0); i < d.Nx; i++) {
    for (int j(0); j < d.Ny; j++) {
      for (int k(0); k < d.Nz; k++) {
        for (int var(0); var < d.Ncons; var++) {
          d.cons[d.id(var, i, j, k)] = 3.1415926;
        }
      }
    }
  }
  // Determine source
  model.sourceTerm(d.cons, d.prims, d.aux, d.source);
  for (int i(0); i < d.Nx; i++) {
    for (int j(0); j < d.Ny; j++) {
      for (int k(0); k < d.Nz; k++) {
        for (int var(0); var < d.Ncons; var++) {
          if (var == 8) EXPECT_EQ(d.source[d.id(var, i, j, k)], -3.1415926 / (0.25));
          else EXPECT_EQ(d.source[d.id(var, i, j, k)], 0);
        }
      }
    }
  }
}



/* ######################### Test getPrimVars transform ########################*/

TEST(SRMHD, Prims2Cons2Prims)
{
  const double tol = 1.49011612e-8;   // Tolerance of rootfinder
  Data d(10, 10, 0, 0, 1, 0, 1, 0, 1, 1.0);
  Data d2(10, 10, 0, 0, 1, 0, 1, 0, 1, 1.0);
  SRMHD model(&d);
  SRMHD model2(&d2);
  Simulation sim(&d);
  Simulation sim2(&d2);
  OTVortexSingleFluid init(&d);
  OTVortexSingleFluid init2(&d2);

  model2.primsToAll(d2.cons, d2.prims, d2.aux);
  model.primsToAll(d.cons, d.prims, d.aux);



  model2.getPrimitiveVars(d2.cons, d2.prims, d2.aux);


  for (int var(0); var < d.Nprims; var++) {
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int k(0); k < d.Nz; k++) {
          EXPECT_NEAR(d.prims[d.id(var, i, j, k)], d2.prims[d.id(var, i, j, k)], tol);
        }
      }
    }
  }
  for (int var(0); var < d.Naux; var++) {
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int k(0); k < d.Nz; k++) {
          EXPECT_NEAR(d.aux[d.id(var, i, j, k)], d2.aux[d.id(var, i, j, k)], tol);
        }
      }
    }
  }

  // Set all d2 prims slightly off so rootfind has to do something
  for (int var(0); var < d.Nprims; var++) {
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int k(0); k < d.Nz; k++) {
          d2.prims[d.id(var, i, j, k)] *= 0.9;
        }
      }
    }
  }
  for (int var(0); var < d.Naux; var++) {
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int k(0); k < d.Nz; k++) {
          d2.aux[d.id(var, i, j, k)] *= 0.9;
        }
      }
    }
  }
  // Solve and re-check
  model2.getPrimitiveVars(d2.cons, d2.prims, d2.aux);

  for (int var(0); var < d.Nprims; var++) {
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int k(0); k < d.Nz; k++) {
          EXPECT_NEAR(d.prims[d.id(var, i, j, k)], d2.prims[d.id(var, i, j, k)], tol);
        }
      }
    }
  }
  for (int var(0); var < d.Naux; var++) {
    for (int i(0); i < d.Nx; i++) {
      for (int j(0); j < d.Ny; j++) {
        for (int k(0); k < d.Nz; k++) {
          EXPECT_NEAR(d.aux[d.id(var, i, j, k)], d2.aux[d.id(var, i, j, k)], tol);
        }
      }
    }
  }

}





/* ######################### Test prims2all transform ########################*/

TEST(SRMHD, PrimsToAll)
{
  // Set up
  Data d(10, 10, 10, 0, 1, 0, 1, 0, 1, 1.0);
  SRMHD model(&d);
  Simulation sim(&d);
  InitialFunc init(&d);

  // Set cons and aux vars to rubbish so we know they have changed, and
  // ser all primitive variables to (nearly) zero: 0 will give zero-division error
  for (int i(0); i < d.Nx; i++) {
    for (int j(0); j < d.Ny; j++) {
      for (int k(0); k < d.Nz; k++) {
        for (int var(0); var < d.Nprims; var++) {
          d.prims[d.id(var, i, j, k)] = 1e-16;
        }
        for (int var(0); var < d.Ncons; var++) {
          d.cons[d.id(var, i, j, k)] = 3.1415926;
        }
        for (int var(0); var < d.Naux; var++) {
          d.aux[d.id(var, i, j, k)] = 2.718281828;
        }
      }
    }
  }

  // Quick check that worked
  EXPECT_EQ(d.cons[d.id(6, 2, 5, 5)], 3.1415926);
  EXPECT_EQ(d.aux[d.id(7, 2, 5, 5)], 2.718281828);
  EXPECT_EQ(d.prims[d.id(7, 8, 4, 5)], 1e-16);

  // Apply conversion and check all cons and aux are zero except h and W
  model.primsToAll(d.cons, d.prims, d.aux);

  for (int i(0); i < d.Nx; i++) {
    for (int j(0); j < d.Ny; j++) {
      for (int k(0); k < d.Nz; k++) {
        // Conserved
        for (int var(0); var < d.Ncons; var++) {
          EXPECT_NEAR(d.cons[d.id(var, i, j, k)], 0.0, 1e-15);
        }
        // Auxiliary
        for (int var(0); var < d.Naux; var++) {
          if (var == 0) EXPECT_EQ(d.aux[d.id(var, i, j, k)], 3.5);
          else if (var == 1) EXPECT_NEAR(d.aux[d.id(var, i, j, k)], 1.0, 1e-15);
          else if (var == 2) EXPECT_EQ(d.aux[d.id(var, i, j, k)], 1.5);
          else if (var == 3) EXPECT_EQ(d.aux[d.id(var, i, j, k)], sqrt(10.0/21.0));
          else EXPECT_NEAR(d.aux[d.id(var, i, j, k)], 0.0, 1e-15);
        }
      }
    }
  }
}
