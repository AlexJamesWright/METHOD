#include "RK2.h"
#include <omp.h>
#include <iostream>
#include <cstdio>

#include "nvToolsExtCuda.h"
#include "nvToolsExtCudaRt.h"


// Macro for getting array index
#define ID(variable, idx, jdx, kdx) ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx)*(d->Ny)*(d->Nz) + (jdx)*(d->Nz) + (kdx))

void RK2::step(double * cons, double * prims, double * aux, double dt)
{

  nvtxRangeId_t profile_id1 = nvtxRangeStartA("RK2 step");

  // Syntax
  Data * d(this->data);

  // Get timestep
  if (dt <= 0) (dt=d->dt);

  // Need some work arrays
  double *p1cons, *p1prims, *p1aux, *args1, *args2;

  int Ntot(d->Nx * d->Ny * d->Nz);

  hipHostAlloc((void **)&p1cons, sizeof(double) * Ntot * d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&p1prims, sizeof(double) * Ntot * d->Nprims,
                hipHostMallocPortable);
  hipHostAlloc((void **)&p1aux, sizeof(double) * Ntot * d->Naux,
                hipHostMallocPortable);
  hipHostAlloc((void **)&args1, sizeof(double) * Ntot * d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&args2, sizeof(double) * Ntot * d->Ncons,
                hipHostMallocPortable);

  nvtxRangeId_t profile_id2 = nvtxRangeStartA("cons2prims");
  // Cons2prims conversion for p1 estimate stage requires old values to start
  // the rootfind
  #pragma omp parallel for
  for (int i=0; i < d->Nx; i++) {
    #pragma omp parallel for
    for (int j=0; j < d->Ny; j++) {
      #pragma omp parallel for
      for (int k=0; k < d->Nz; k++) {
        #pragma omp parallel for
        for (int var=0; var < d->Naux; var++) {
          p1aux[ID(var, i, j, k)] = aux[ID(var, i, j, k)];
        }
        #pragma omp parallel for
        for (int var=0; var < d->Nprims; var++) {
          p1prims[ID(var, i, j, k)] = prims[ID(var, i, j, k)];
        }
      }
    }
  }
  nvtxRangeEnd(profile_id2);

  nvtxRangeId_t profile_id3 = nvtxRangeStartA("fluxMethod");
  // Get first approximation of flux contribution
  this->fluxMethod->F(cons, prims, aux, d->f, args1);
  nvtxRangeEnd(profile_id3);

  nvtxRangeId_t profile_id4 = nvtxRangeStartA("cons2prims 2");
  // First stage approximation
  #pragma omp parallel for
   for (int var=0; var < d->Ncons; var++) {
     #pragma omp parallel for
     for (int i=0; i < d->Nx; i++) {
       #pragma omp parallel for
       for (int j=0; j < d->Ny; j++) {
         #pragma omp parallel for
         for (int k=0; k < d->Nz; k++) {
           p1cons[ID(var, i, j, k)] = cons[ID(var, i, j, k)] - dt * args1[ID(var, i, j, k)];
         }
       }
     }
   }
   // Apply boundary conditions and get primitive and aux vars for p1
   try {
     this->model->getPrimitiveVars(p1cons, p1prims, p1aux);
   }
   catch (const std::exception& e) {
     printf("RK2 (stage 1) raises exception with following message:\n%s\n", e.what());
     throw e;
   }
   nvtxRangeEnd(profile_id4);

   nvtxRangeId_t profile_id5 = nvtxRangeStartA("bcs");
   this->bcs->apply(p1cons, p1prims, p1aux);
   nvtxRangeEnd(profile_id5);

   nvtxRangeId_t profile_id6 = nvtxRangeStartA("fluxMethod 2");
   // Get second approximation of flux contribution
   this->fluxMethod->F(p1cons, p1prims, p1aux, d->f, args2);
   nvtxRangeEnd(profile_id6);

   nvtxRangeId_t profile_id7 = nvtxRangeStartA("construct solution");
   // Construct solution
   #pragma omp parallel for
   for (int var=0; var < d->Ncons; var++) {
     #pragma omp parallel for
     for (int i=0; i < d->Nx; i++) {
       #pragma omp parallel for
       for (int j=0; j < d->Ny; j++) {
         #pragma omp parallel for
         for (int k=0; k < d->Nz; k++) {
           cons[ID(var, i, j, k)] = 0.5 * (cons[ID(var, i, j, k)] + p1cons[ID(var, i, j, k)] -
                                       dt * args2[ID(var, i, j, k)]);
         }
       }
     }
   }
   nvtxRangeEnd(profile_id7);

   nvtxRangeId_t profile_id8 = nvtxRangeStartA("get prims");
   // Determine new prim and aux variables
   try {
     this->model->getPrimitiveVars(cons, prims, aux);
   }
   catch (const std::exception& e) {
     printf("RK2 (corrector) raises exception with following message:\n%s\n", e.what());
     throw e;
   }
   nvtxRangeEnd(profile_id8);

   nvtxRangeId_t profile_id9 = nvtxRangeStartA("bcs 2");
   // Apply boundary conditions
   this->bcs->apply(cons, prims, aux);
   nvtxRangeEnd(profile_id9);

   // Free arrays
   hipHostFree(p1cons);
   hipHostFree(p1prims);
   hipHostFree(p1aux);
   hipHostFree(args1);
   hipHostFree(args2);

   nvtxRangeEnd(profile_id1);
}
