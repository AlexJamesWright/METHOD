#include "C2PArgs.h"
#include <cstdio>
#include "cudaErrorCheck.h"


C2PArgs::C2PArgs(Data * data) : data(data)
{
  // Syntax
  Data * d(this->data);

  // Determine the memory required for one cell
  cellMem = (d->Ncons + d->Nprims + d->Naux) * sizeof(double);

  tpb = d->tpb;
  bpg = d->bpg;
  streamWidth = tpb * bpg;
  Nstreams = d->Nstreams;


  // Device arrays for each stream
  cons_d = new double*[Nstreams];
  prims_d = new double*[Nstreams];
  aux_d = new double*[Nstreams];
  guess_d = new double*[Nstreams];
  // Host arrays
  gpuErrchk( hipHostAlloc((void **)&cons_h, d->Ncons * d->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&prims_h, d->Nprims * d->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&aux_h, d->Naux * d->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&guess_h, d->Ncells * sizeof(double), hipHostMallocPortable) );



  for (int i(0); i < Nstreams; i++) {
    gpuErrchk( hipMalloc((void **)&cons_d[i], d->Ncons * streamWidth * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&prims_d[i], d->Nprims * streamWidth * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&aux_d[i], d->Naux * streamWidth * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&guess_d[i], streamWidth * sizeof(double)) );
  }

  // Create streams
  stream = new hipStream_t[Nstreams];
  for (int i(0); i<Nstreams; i++) {
    gpuErrchk( hipStreamCreate(&stream[i]) );
  }

}

C2PArgs::~C2PArgs()
{
  for (int i(0); i < Nstreams; i++) {
    gpuErrchk( hipFree(cons_d[i]) );
    gpuErrchk( hipFree(prims_d[i]) );
    gpuErrchk( hipFree(aux_d[i]) );
    gpuErrchk( hipFree(guess_d[i]) );
  }
  gpuErrchk( hipHostFree(cons_h) );
  gpuErrchk( hipHostFree(prims_h) );
  gpuErrchk( hipHostFree(aux_h) );
  gpuErrchk( hipHostFree(guess_h) );
}
