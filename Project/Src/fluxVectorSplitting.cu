#include "fluxVectorSplitting.h"

void FVS::fluxReconstruction(double * cons, double * prims, double * aux, double * f, double * frecon, int dir)
{
  // Syntax
  Data * d(this->data);

  // Order of weno scheme
  int order(2);

  // Wave speed
  double alpha;
  if (dir == 0) alpha = d->alphaX;
  else if (dir == 1) alpha = d->alphaY;
  else alpha = d->alphaZ;

  // Up and downwind fluxes
  double *fplus, *fminus;
  hipHostAlloc((void **)&fplus, sizeof(double) * d->Ncons * d->Nx * d->Ny * d->Nz,
                hipHostMallocPortable);
  hipHostAlloc((void **)&fminus, sizeof(double) * d->Ncons * d->Nx * d->Ny * d->Nz,
                  hipHostMallocPortable);

  // Get flux vector
  this->model->fluxVector(cons, prims, aux, f, dir);

  // Lax-Friedrichs approximation of flux
  for (int var(0); var < d->Ncons; var++) {
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        for (int k(0); k < d->Nz; k++) {
          fplus[d->id(var, i, j, k)] = 0.5 * (f[d->id(var, i, j, k)] + alpha * cons[d->id(var, i, j, k)]);
          fminus[d->id(var, i, j, k)] = 0.5 * (f[d->id(var, i, j, k)] - alpha * cons[d->id(var, i, j, k)]);
        }
      }
    }
  }

  // Reconstruct to determine the flux at the cell face and compute difference
  if (dir == 0) { // x-direction
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx; i++) {
        for (int j(0); j < d->Ny; j++) {
          for (int k(0); k < d->Nz; k++) {
            if (i >= order && i < d->Nx-order) {
              frecon[d->id(var, i, j, k)] = weno3_upwind(fplus[d->id(var, i-order, j, k)],
                                                         fplus[d->id(var, i-order+1, j, k)],
                                                         fplus[d->id(var, i-order+2, j, k)]) +
                                            weno3_upwind(fminus[d->id(var, i+order-1, j, k)],
                                                         fminus[d->id(var, i+order-2, j, k)],
                                                         fminus[d->id(var, i+order-3, j, k)]);
            }
            else {
              frecon[d->id(var, i, j, k)] = 0.0;
            }
          }
        }
      }
    }
  }
  else if (dir == 1) { // y-direction
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx; i++) {
        for (int j(0); j < d->Ny; j++) {
          for (int k(0); k < d->Nz; k++) {
            if (j >= order && j < d->Ny-order) {
              frecon[d->id(var, i, j, k)] = weno3_upwind(fplus[d->id(var, i, j-order, k)],
                                                         fplus[d->id(var, i, j-order+1, k)],
                                                         fplus[d->id(var, i, j-order+2, k)]) +
                                            weno3_upwind(fminus[d->id(var, i, j+order-1, k)],
                                                         fminus[d->id(var, i, j+order-2, k)],
                                                         fminus[d->id(var, i, j+order-3, k)]);
            }
            else {
              frecon[d->id(var, i, j, k)] = 0.0;
            }
          }
        }
      }
    }
  }
  else { // z-direction
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx; i++) {
        for (int j(0); j < d->Ny; j++) {
          for (int k(0); k < d->Nz; k++) {
            if (k >= order && k < d->Nz-order) {
              frecon[d->id(var, i, j, k)] = weno3_upwind(fplus[d->id(var, i, j, k-order)],
                                                         fplus[d->id(var, i, j, k-order+1)],
                                                         fplus[d->id(var, i, j, k-order+2)]) +
                                            weno3_upwind(fminus[d->id(var, i, j, k+order-1)],
                                                         fminus[d->id(var, i, j, k+order-2)],
                                                         fminus[d->id(var, i, j, k+order-3)]);
            }
            else {
              frecon[d->id(var, i, j, k)] = 0.0;
            }
          }
        }
      }
    }
  }
  // Free arrays
  hipHostFree(fplus);
  hipHostFree(fminus);
}

void FVS::F(double * cons, double * prims, double * aux, double * f, double * fnet)
{
  // Syntax
  Data * d(this->data);

  // Reconstructed fluxes in x, y, z direction
  double *fx, *fy, *fz;

  // 3D domain, loop over all cells determining the net flux
  if (d->Ny > 1 && d->Nz > 1) {
    hipHostAlloc((void **)&fx, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable);
    hipHostAlloc((void **)&fy, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable);
    hipHostAlloc((void **)&fz, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable);
    // Determine flux vectors
    this->fluxReconstruction(cons, prims, aux, f, fx, 0);
    this->fluxReconstruction(cons, prims, aux, f, fy, 1);
    this->fluxReconstruction(cons, prims, aux, f, fz, 2);
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
        for (int j(0); j < d->Ny-1; j++) {
          for (int k(0); k < d->Nz-1; k++) {
            fnet[d->id(var, i, j, k)] = (fx[d->id(var, i+1, j, k)] / d->dx - fx[d->id(var, i, j, k)] / d->dx) +
                                        (fy[d->id(var, i, j+1, k)] / d->dy - fy[d->id(var, i, j, k)] / d->dy) +
                                        (fz[d->id(var, i, j, k+1)] / d->dz - fz[d->id(var, i, j, k)] / d->dz);
          }
        }
      }
    }
    hipHostFree(fx);
    hipHostFree(fy);
    hipHostFree(fz);
  }


  // 2D domain, loop over x- and y-directions determining the net flux
  else if (d->Ny > 1) {
    hipHostAlloc((void **)&fx, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable);
    hipHostAlloc((void **)&fy, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable);
    this->fluxReconstruction(cons, prims, aux, f, fx, 0);
    this->fluxReconstruction(cons, prims, aux, f, fy, 1);
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
        for (int j(0); j < d->Ny-1; j++) {
          fnet[d->id(var, i, j, 0)] = (fx[d->id(var, i+1, j, 0)] / d->dx - fx[d->id(var, i, j, 0)] / d->dx) +
                                      (fy[d->id(var, i, j+1, 0)] / d->dy - fy[d->id(var, i, j, 0)] / d->dy);

        }
      }
    }
    hipHostFree(fx);
    hipHostFree(fy);

  }


  // Otherwise, domain is 1D only loop over x direction
  else {
    hipHostAlloc((void **)&fx, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable);
    this->fluxReconstruction(cons, prims, aux, f, fx, 0);
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
          fnet[d->id(var, i, 0, 0)] = (fx[d->id(var, i+1, 0, 0)] / d->dx - fx[d->id(var, i, 0, 0)] / d->dx);
      }
    }
    hipHostFree(fx);
  }
}
