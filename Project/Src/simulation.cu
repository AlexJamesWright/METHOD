#include "simulation.h"
#include "cudaErrorCheck.h"

Simulation::Simulation(Data * data) : data(data)
{
  // Allocate memory for state arrays
  int Ntot(this->data->Nx * this->data->Ny);

  gpuErrchk( hipHostAlloc((void **)&this->data->cons,
                sizeof(double) * Ntot * this->data->Ncons,
                hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&this->data->f,
                sizeof(double) * Ntot * this->data->Ncons,
                hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&this->data->fnet,
                sizeof(double) * Ntot * this->data->Ncons,
                hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&this->data->source,
                sizeof(double) * Ntot * this->data->Ncons,
                hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&this->data->prims,
                sizeof(double) * Ntot * this->data->Nprims,
                hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&this->data->aux,
                sizeof(double) * Ntot * this->data->Naux,
                hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&this->data->x,
                sizeof(double) * Ntot,
                hipHostMallocPortable) );
}

Simulation::~Simulation()
{
  // Need to free arrays
  gpuErrchk( hipHostFree(this->data->cons) );
  gpuErrchk( hipHostFree(this->data->f) );
  gpuErrchk( hipHostFree(this->data->fnet) );
  gpuErrchk( hipHostFree(this->data->source) );
  gpuErrchk( hipHostFree(this->data->prims) );
  gpuErrchk( hipHostFree(this->data->aux) );
  gpuErrchk( hipHostFree(this->data->x) );
}
