#include "rkSplit.h"
#include <stdio.h>

void RKSplit::step()
{
  // Syntax
  Data * d(this->data);

  // Need some work arrays
  double *p1, *args1, *args2;

  hipHostAlloc((void **)&p1, sizeof(double) * d->Nx * d->Ny * d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&args1, sizeof(double) * d->Nx * d->Ny * d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&args2, sizeof(double) * d->Nx * d->Ny * d->Ncons,
                hipHostMallocPortable);


  //   Im not entirely convinced this is the correct way of doing things
  // but its certainly slightly less effort and its what we've done in the past
  // so will do for now. Re-visit this.
  //   I've a sneeking suspicion we need to find the primitive vars for each
  // of the stages estimates.

  // Get first approximation of flux contribution
  this->model->F(d->cons, d->prims, d->aux, d->f, args1);

  // First stage approximation
   for (int var(0); var < d->Ncons; var++) {
     for (int i(0); i < d->Nx; i++) {
       for (int j(0); j < d->Ny; j++) {
         p1[d->id(var, i, j)] = d->cons[d->id(var, i, j)] - d->dt * args1[d->id(var, i, j)];
       }
     }
   }

   // Apply boundary conditions
   this->bc->apply(p1);

   // Get second approximation of flux contribution
   this->model->F(p1, d->prims, d->aux, d->f, args2);

   // Construct solution
   for (int var(0); var < d->Ncons; var++) {
     for (int i(0); i < d->Nx; i++) {
       for (int j(0); j < d->Ny; j++) {
         d->cons[d->id(var, i, j)] = 0.5 * (d->cons[d->id(var, i, j)] + p1[d->id(var, i, j)] -
                                         d->dt * args2[d->id(var, i, j)]);
       }
     }
   }

   // Add source contribution
   this->model->sourceTerm(d->cons, d->prims, d->aux, d->source);
   for (int var(0); var < d->Ncons; var++) {
     for (int i(0); i < d->Nx; i++) {
       for (int j(0); j < d->Ny; j++) {
         d->cons[d->id(var, i, j)] += d->dt * d->source[d->id(var, i, j)];
       }
     }
   }

   // Apply boundary conditions
   this->bc->apply(d->cons);


   // Determine new prim and aux variables
   this->model->getPrimitiveVars(d->cons, d->prims, d->aux);


   // Free arrays
   hipHostFree(p1);
   hipHostFree(args1);
   hipHostFree(args2);

}
