#include "hip/hip_runtime.h"
//! Two-Fluid ElectroMagnetoHydroDynamics model
/*!
    Script contains the function definitions for the two fluid model of Amano 2016
  accompanied by the divergence cleaning method to enforce the contraints set by
  Maxwell's equations.
*/

#include "twoFluidEMHD.h"
#include "weno.h"
#include <cmath>
#include <cstdio>

// Declare cons2prims residual function and Newton Solver
static double residual(const double, const double, const double, const double, double);
static void newton(double *, const double, const double, const double, double);

TwoFluidEMHD::TwoFluidEMHD() : Model()
{
  this->Ncons = 12;
  this->Nprims = 16;
  this->Naux = 38;
}

TwoFluidEMHD::TwoFluidEMHD(Data * data) : Model(data)
{
  // Syntax
  Data * d(this->data);

  this->Ncons = d->Ncons = 18;
  this->Nprims = d->Nprims = 16;
  this->Naux = d->Naux = 35;

  d->consLabels.push_back("D");       d->consLabels.push_back("Sx");
  d->consLabels.push_back("Sy");      d->consLabels.push_back("Sz");
  d->consLabels.push_back("Tau");     d->consLabels.push_back("Dbar");
  d->consLabels.push_back("Sbarx");   d->consLabels.push_back("Sbary");
  d->consLabels.push_back("Sbarz");   d->consLabels.push_back("taubar");
  d->consLabels.push_back("Bx");      d->consLabels.push_back("By");
  d->consLabels.push_back("Bz");      d->consLabels.push_back("Ex");
  d->consLabels.push_back("Ey");      d->consLabels.push_back("Ez");
  d->consLabels.push_back("psi");     d->consLabels.push_back("phi");

  d->primsLabels.push_back("rho1");   d->primsLabels.push_back("vx1");
  d->primsLabels.push_back("vy1");    d->primsLabels.push_back("vz1");
  d->primsLabels.push_back("p1");     d->primsLabels.push_back("rho2");
  d->primsLabels.push_back("vx2");    d->primsLabels.push_back("vy2");
  d->primsLabels.push_back("vz2");    d->primsLabels.push_back("p2");
  d->primsLabels.push_back("Bx");     d->primsLabels.push_back("By");
  d->primsLabels.push_back("Bz");     d->primsLabels.push_back("Ex");
  d->primsLabels.push_back("Ey");     d->primsLabels.push_back("Ez");

  d->auxLabels.push_back("h1");       d->auxLabels.push_back("W1");
  d->auxLabels.push_back("e1");       d->auxLabels.push_back("vsq1");
  d->auxLabels.push_back("Z1");       d->auxLabels.push_back("D1");
  d->auxLabels.push_back("Stildex1"); d->auxLabels.push_back("Stildey1");
  d->auxLabels.push_back("Stildez1"); d->auxLabels.push_back("tauTilde1");
  d->auxLabels.push_back("h2");       d->auxLabels.push_back("W2");
  d->auxLabels.push_back("e2");       d->auxLabels.push_back("vsq2");
  d->auxLabels.push_back("Z2");       d->auxLabels.push_back("D2");
  d->auxLabels.push_back("Stildex2"); d->auxLabels.push_back("Stildey2");
  d->auxLabels.push_back("Stildez2"); d->auxLabels.push_back("tauTilde2");
  d->auxLabels.push_back("Bsq");      d->auxLabels.push_back("Esq");
  d->auxLabels.push_back("Jx");       d->auxLabels.push_back("Jy");
  d->auxLabels.push_back("Jz");       d->auxLabels.push_back("Stildex");
  d->auxLabels.push_back("Stildey");  d->auxLabels.push_back("Stilfdez");
  d->auxLabels.push_back("tauTilde"); d->auxLabels.push_back("rhoCh");
  d->auxLabels.push_back("rhoCh0");   d->auxLabels.push_back("ux");
  d->auxLabels.push_back("uy");       d->auxLabels.push_back("uz");
  d->auxLabels.push_back("W");
}

void TwoFluidEMHD::fluxFunc(double *cons, double *prims, double *aux, double *f, double *fnet, const int dir)
{
  // Syntax
  Data * d(this->data);

  // up and downwind fluxes
  double *fplus, *fminus;

  hipHostAlloc((void **)&fplus, sizeof(double)*d->Nx*d->Ny*d->Nz*d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&fminus, sizeof(double)*d->Nx*d->Ny*d->Nz*d->Ncons,
                hipHostMallocPortable);

  // Wave speed
  double alpha;
  if (dir == 0) alpha = d->alphaX;
  else if (dir == 1) alpha = d->alphaY;
  else alpha = d->alphaZ;

  // Order of weno scheme
  int order(2);

  // Generate flux vector
  // Fx: flux in x-direction
  if (dir == 0) {
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        for (int k(0); k < d->Nz; k++) {
          // D
          f[d->id(0, i, j, k)] = aux[d->id(5, i, j, k)] * prims[d->id(1, i, j, k)] +
                                 aux[d->id(15, i, j, k)] * prims[d->id(6, i, j, k)];
          // Sx, Sy, Sx
          f[d->id(1, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(1, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(6, i, j, k)] * prims[d->id(6, i, j, k)] +
                                 prims[d->id(4, i, j, k)] + prims[d->id(9, i, j, k)] -
                                 (cons[d->id(13, i, j, k)] * cons[d->id(13 ,i, j, k)] +
                                 cons[d->id(10, i, j, k)] * cons[d->id(10, i, j, k)]) +
                                 (aux[d->id(20, i, j, k)] + aux[d->id(21, i, j, k)]) * 0.5;
          f[d->id(2, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(6, i, j, k)] * prims[d->id(7, i, j, k)] -
                                 (cons[d->id(13, i, j, k)] * cons[d->id(14, i, j, k)] +
                                 cons[d->id(10, i, j, k)] * cons[d->id(11, i, j, k)]);
          f[d->id(3, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(6, i, j, k)] * prims[d->id(8, i, j, k)] -
                                 (cons[d->id(13, i, j, k)] * cons[d->id(15, i, j, k)] +
                                 cons[d->id(10, i, j, k)] * cons[d->id(12, i, j, k)]);
          // Tau
          f[d->id(4, i, j, k)] = cons[d->id(1, i, j, k)] - (aux[d->id(5, i, j, k)] *
                                 prims[d->id(1, i, j, k)] + aux[d->id(15, i, j, k)] *
                                 prims[d->id(6, i, j, k)]);
          // Dbar
          f[d->id(5, i, j, k)] = d->mu1 * aux[d->id(5, i, j, k)] * prims[d->id(1, i, j, k)] +
                                 d->mu2 * aux[d->id(15, i, j, k)] * prims[d->id(6, i, j, k)];
          // Sbarx, Sbary, Sbarz
          f[d->id(6, i, j, k)] = d->mu1 * (aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(1, i, j, k)] + prims[d->id(4, i, j, k)]) +
                                 d->mu2 * (aux[d->id(14, i, j, k)] * prims[d->id(6, i, j, k)] *
                                 prims[d->id(6, i, j, k)] + prims[d->id(9, i, j, k)]);
          f[d->id(7, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + d->mu2 * aux[d->id(14, i, j, k)] *
                                 prims[d->id(6, i, j, k)] * prims[d->id(7, i, j, k)];
          f[d->id(8, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + d->mu2 * aux[d->id(14, i, j, k)] *
                                 prims[d->id(6, i, j, k)] * prims[d->id(8, i, j, k)];
          // tauBar
          f[d->id(9, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] +
                                 d->mu2 * aux[d->id(14, i, j, k)] * prims[d->id(6, i, j, k)] -
                                 (d->mu1 * aux[d->id(5, i, j, k)] * prims[d->id(1, i, j, k)] +
                                 d->mu2 * aux[d->id(15, i, j, k)] * prims[d->id(6, i, j, k)]);
          // Bx, By, Bz
          f[d->id(10, i, j, k)] = cons[d->id(17, i, j, k)];
          f[d->id(11, i, j, k)] = - cons[d->id(15, i, j, k)];
          f[d->id(12, i, j, k)] = cons[d->id(14, i, j, k)];
          // Ex, Ey, Ez
          f[d->id(13, i, j, k)] = cons[d->id(16, i, j, k)];
          f[d->id(14, i, j, k)] = cons[d->id(12, i, j, k)];
          f[d->id(15, i, j, k)] = - cons[d->id(11, i, j, k)];
          // Psi, Phi
          f[d->id(16, i, j, k)] = cons[d->id(13, i, j, k)];
          f[d->id(17, i, j, k)] = cons[d->id(10, i, j, k)];
        }
      }
    }
  }
  // Fy: flux in y-direction
  else if (dir == 1) {
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        for (int k(0); k < d->Nz; k++) {
          // D
          f[d->id(0, i, j, k)] = aux[d->id(5, i, j, k)] * prims[d->id(2, i, j, k)] +
                                 aux[d->id(15, i, j, k)] * prims[d->id(7, i, j, k)];
          // Sx, Sy, Sx
          f[d->id(1, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(6, i, j, k)] * prims[d->id(7, i, j, k)] -
                                 (cons[d->id(13, i, j, k)] * cons[d->id(14 ,i, j, k)] +
                                 cons[d->id(10, i, j, k)] * cons[d->id(11, i, j, k)]);
          f[d->id(2, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(2, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(7, i, j, k)] * prims[d->id(7, i, j, k)] +
                                 prims[d->id(4, i, j, k)] + prims[d->id(9, i, j, k)] -
                                 (cons[d->id(14, i, j, k)] * cons[d->id(14, i, j, k)] +
                                 cons[d->id(11, i, j, k)] * cons[d->id(11, i, j, k)]) +
                                 (aux[d->id(20, i, j, k)] + aux[d->id(21, i, j, k)]) * 0.5;
          f[d->id(3, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(3, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(8, i, j, k)] * prims[d->id(7, i, j, k)] -
                                 (cons[d->id(15, i, j, k)] * cons[d->id(14, i, j, k)] +
                                 cons[d->id(12, i, j, k)] * cons[d->id(11, i, j, k)]);
          // Tau
          f[d->id(4, i, j, k)] = cons[d->id(2, i, j, k)] - (aux[d->id(5, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + aux[d->id(15, i, j, k)] *
                                 prims[d->id(7, i, j, k)]);
          // Dbar
          f[d->id(5, i, j, k)] = d->mu1 * aux[d->id(5, i, j, k)] * prims[d->id(2, i, j, k)] +
                                 d->mu2 * aux[d->id(15, i, j, k)] * prims[d->id(7, i, j, k)];
          // Sbarx, Sbary, Sbarz
          f[d->id(6, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(2, i, j, k)] *
                                 prims[d->id(1, i, j, k)] + d->mu2 * aux[d->id(14, i, j, k)] *
                                 prims[d->id(7, i, j, k)] * prims[d->id(6, i, j, k)] ;
          f[d->id(7, i, j, k)] = d->mu1 * (aux[d->id(4, i, j, k)] * prims[d->id(2, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + prims[d->id(4, i, j, k)]) +
                                 d->mu2 * (aux[d->id(14, i, j, k)] * prims[d->id(7, i, j, k)] *
                                 prims[d->id(7, i, j, k)] + prims[d->id(9, i, j, k)]);
          f[d->id(8, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(2, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + d->mu2 * aux[d->id(14, i, j, k)] *
                                 prims[d->id(7, i, j, k)] * prims[d->id(8, i, j, k)];
          // tauBar
          f[d->id(9, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(2, i, j, k)] +
                                 d->mu2 * aux[d->id(14, i, j, k)] * prims[d->id(7, i, j, k)] -
                                 (d->mu1 * aux[d->id(5, i, j, k)] * prims[d->id(2, i, j, k)] +
                                 d->mu2 * aux[d->id(15, i, j, k)] * prims[d->id(7, i, j, k)]);
          // Bx, By, Bz
          f[d->id(10, i, j, k)] = cons[d->id(15, i, j, k)];
          f[d->id(11, i, j, k)] = cons[d->id(17, i, j, k)];
          f[d->id(12, i, j, k)] = - cons[d->id(13, i, j, k)];
          // Ex, Ey, Ez
          f[d->id(13, i, j, k)] = - cons[d->id(12, i, j, k)];
          f[d->id(14, i, j, k)] = cons[d->id(16, i, j, k)];
          f[d->id(15, i, j, k)] = cons[d->id(10, i, j, k)];
          // Psi, Phi
          f[d->id(16, i, j, k)] = cons[d->id(14, i, j, k)];
          f[d->id(17, i, j, k)] = cons[d->id(11, i, j, k)];
        }
      }
    }
  }
  // Fz: flux in z-direction
  else {
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        for (int k(0); k < d->Nz; k++) {
          // D
          f[d->id(0, i, j, k)] = aux[d->id(5, i, j, k)] * prims[d->id(3, i, j, k)] +
                                 aux[d->id(15, i, j, k)] * prims[d->id(8, i, j, k)];
          // Sx, Sy, Sx
          f[d->id(1, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(1, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(6, i, j, k)] * prims[d->id(8, i, j, k)] -
                                 (cons[d->id(13, i, j, k)] * cons[d->id(14 ,i, j, k)] +
                                 cons[d->id(10, i, j, k)] * cons[d->id(11, i, j, k)]);
          f[d->id(2, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(2, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(7, i, j, k)] * prims[d->id(8, i, j, k)] -
                                 (cons[d->id(14, i, j, k)] * cons[d->id(14, i, j, k)] +
                                 cons[d->id(11, i, j, k)] * cons[d->id(11, i, j, k)]);
          f[d->id(3, i, j, k)] = aux[d->id(4, i, j, k)] * prims[d->id(3, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + aux[d->id(14, i, j, k)] *
                                 prims[d->id(8, i, j, k)] * prims[d->id(8, i, j, k)] +
                                 prims[d->id(4, i, j, k)] + prims[d->id(9, i, j, k)] -
                                 (cons[d->id(15, i, j, k)] * cons[d->id(14, i, j, k)] +
                                 cons[d->id(12, i, j, k)] * cons[d->id(11, i, j, k)]) +
                                 (aux[d->id(20, i, j, k)] + aux[d->id(21, i, j, k)]) * 0.5;
          // Tau
          f[d->id(4, i, j, k)] = cons[d->id(3, i, j, k)] - (aux[d->id(5, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + aux[d->id(15, i, j, k)] *
                                 prims[d->id(8, i, j, k)]);
          // Dbar
          f[d->id(5, i, j, k)] = d->mu1 * aux[d->id(5, i, j, k)] * prims[d->id(2, i, j, k)] +
                                 d->mu2 * aux[d->id(15, i, j, k)] * prims[d->id(7, i, j, k)];
          // Sbarx, Sbary, Sbarz
          f[d->id(6, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(3, i, j, k)] *
                                 prims[d->id(1, i, j, k)] + d->mu2 * aux[d->id(14, i, j, k)] *
                                 prims[d->id(8, i, j, k)] * prims[d->id(6, i, j, k)] ;
          f[d->id(7, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(3, i, j, k)] *
                                 prims[d->id(2, i, j, k)] + d->mu2 * aux[d->id(14, i, j, k)] *
                                 prims[d->id(8, i, j, k)] * prims[d->id(7, i, j, k)];
          f[d->id(8, i, j, k)] = d->mu1 * (aux[d->id(4, i, j, k)] * prims[d->id(3, i, j, k)] *
                                 prims[d->id(3, i, j, k)] + prims[d->id(4, i, j, k)]) +
                                 d->mu2 * (aux[d->id(14, i, j, k)] * prims[d->id(8, i, j, k)] *
                                 prims[d->id(8, i, j, k)] + prims[d->id(9, i, j, k)]);
          // tauBar
          f[d->id(9, i, j, k)] = d->mu1 * aux[d->id(4, i, j, k)] * prims[d->id(3, i, j, k)] +
                                 d->mu2 * aux[d->id(14, i, j, k)] * prims[d->id(8, i, j, k)] -
                                 (d->mu1 * aux[d->id(5, i, j, k)] * prims[d->id(3, i, j, k)] +
                                 d->mu2 * aux[d->id(15, i, j, k)] * prims[d->id(8, i, j, k)]);
          // Bx, By, Bz
          f[d->id(10, i, j, k)] = - cons[d->id(14, i, j, k)];
          f[d->id(11, i, j, k)] = cons[d->id(13, i, j, k)];
          f[d->id(12, i, j, k)] = cons[d->id(17, i, j, k)];
          // Ex, Ey, Ez
          f[d->id(13, i, j, k)] = cons[d->id(11, i, j, k)];
          f[d->id(14, i, j, k)] = - cons[d->id(10, i, j, k)];
          f[d->id(15, i, j, k)] = cons[d->id(16, i, j, k)];
          // Psi, Phi
          f[d->id(16, i, j, k)] = cons[d->id(15, i, j, k)];
          f[d->id(17, i, j, k)] = cons[d->id(12, i, j, k)];
        }
      } // End k loop
    } // End j loop
  } // End i loop

  // Lax-Friedrichs approximation of flux
  for (int var(0); var < d->Ncons; var++) {
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        for (int k(0); k < d->Nz; k++) {
          fplus[d->id(var, i, j, k)] = 0.5 * (f[d->id(var, i, j, k)] + alpha * cons[d->id(var, i, j, k)]);
          fminus[d->id(var, i, j, k)] = 0.5 * (f[d->id(var, i, j, k)] - alpha * cons[d->id(var, i, j, k)]);
        }
      }
    }
  }


    // Reconstruct to determine the flux at the cell face and compute difference
    if (dir == 0) { // x-direction
      for (int var(0); var < d->Ncons; var++) {
        for (int i(order); i < d->Nx-order; i++) {
          for (int j(0); j < d->Ny; j++) {
            for (int k(0); k < d->Nz; k++) {
              fnet[d->id(var, i, j, k)] = weno3_upwind(fplus[d->id(var, i-order, j, k)],
                                                       fplus[d->id(var, i-order+1, j, k)],
                                                       fplus[d->id(var, i-order+2, j, k)]) +
                                          weno3_upwind(fminus[d->id(var, i+order-1, j, k)],
                                                       fminus[d->id(var, i+order-2, j, k)],
                                                       fminus[d->id(var, i+order-3, j, k)]);
            }
          }
        }
      }
    }
    else if (dir == 1) { // y-direction
      for (int var(0); var < d->Ncons; var++) {
        for (int i(0); i < d->Nx; i++) {
          for (int j(order); j < d->Ny-order; j++) {
            for (int k(0); k < d->Nz; k++) {
              fnet[d->id(var, i, j, k)] = weno3_upwind(fplus[d->id(var, i, j-order, k)],
                                                       fplus[d->id(var, i, j-order+1, k)],
                                                       fplus[d->id(var, i, j-order+2, k)]) +
                                          weno3_upwind(fminus[d->id(var, i, j+order-1, k)],
                                                       fminus[d->id(var, i, j+order-2, k)],
                                                       fminus[d->id(var, i, j+order-3, k)]);
            }
          }
        }
      }
    }
    else { // z-direction
      for (int var(0); var < d->Ncons; var++) {
        for (int i(0); i < d->Nx; i++) {
          for (int j(0); j < d->Ny; j++) {
            for (int k(order); k < d->Nz-order; k++) {
              fnet[d->id(var, i, j, k)] = weno3_upwind(fplus[d->id(var, i, j, k-order)],
                                                       fplus[d->id(var, i, j, k-order+1)],
                                                       fplus[d->id(var, i, j, k-order+2)]) +
                                          weno3_upwind(fminus[d->id(var, i, j, k+order-1)],
                                                       fminus[d->id(var, i, j, k+order-2)],
                                                       fminus[d->id(var, i, j, k+order-3)]);
            }
          }
        }
      }
    }

    // Free arrays
    hipHostFree(fplus);
    hipHostFree(fminus);

}


//! Numerical flux approximation
void TwoFluidEMHD::F(double *cons, double *prims, double *aux, double *f, double *fnet)
{

  // Syntax
  Data * d(this->data);

  double *fx, *fy, *fz;

  hipHostAlloc((void **)&fx, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&fy, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                hipHostMallocPortable);

  // Determine fluxes at cell faces
  this->fluxFunc(cons, prims, aux, f, fx, 0);
  this->fluxFunc(cons, prims, aux, f, fy, 1);

  // If domain is 3D loop over z direction also
  if (d->Nz > 1) {
    hipHostAlloc((void **)&fz, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable);
    this->fluxFunc(cons, prims, aux, f, fz, 2);
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
        for (int j(0); j < d->Ny-1; j++) {
          for (int k(0); k < d->Nz-1; k++) {
            fnet[d->id(var, i, j, k)] = (fx[d->id(var, i+1, j, k)] / d->dx - fx[d->id(var, i, j, k)] / d->dx) +
                                        (fy[d->id(var, i, j+1, k)] / d->dy - fy[d->id(var, i, j, k)] / d->dy) +
                                        (fz[d->id(var, i, j, k+1)] / d->dz - fz[d->id(var, i, j, k)] / d->dz);
          }
        }
      }
    }
    hipHostFree(fz);
  }
  // Otherwise there is only one k cell
  else {
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
        for (int j(0); j < d->Ny-1; j++) {
          fnet[d->id(var, i, j, 0)] = (fx[d->id(var, i+1, j, 0)] / d->dx - fx[d->id(var, i, j, 0)] / d->dx) +
                                      (fy[d->id(var, i, j+1, 0)] / d->dy - fy[d->id(var, i, j, 0)] / d->dy);

        }
      }
    }
  }

  // Free arrays
  hipHostFree(fx);
  hipHostFree(fy);
}

//! Source contribution
void TwoFluidEMHD::sourceTerm(double *cons, double *prims, double *aux, double *source)
{
  // Syntax
  Data * d(this->data);

  for (int i(0); i < this->data->Nx; i++) {
    for (int j(0); j < this->data->Ny; j++) {
      for (int k(0); k < this->data->Nz; k++) {
        for (int var(0); var < this->data->Ncons; var++) {
          source[d->id(0, i, j, k)] = 0;
          source[d->id(1, i, j, k)] = 0;
          source[d->id(2, i, j, k)] = 0;
          source[d->id(3, i, j, k)] = 0;
          source[d->id(4, i, j, k)] = 0;
          source[d->id(5, i, j, k)] = 0;
          source[d->id(6, i, j, k)] = aux[d->id(34, i, j, k)] * cons[d->id(13, i, j, k)] +
                                      (aux[d->id(32, i, j, k)] * cons[d->id(12, i, j, k)] -
                                      aux[d->id(33, i, j, k)] * cons[d->id(11, i, j, k)]) -
                                      (aux[d->id(22, i, j, k)] - aux[d->id(29, i, j, k)] *
                                      aux[d->id(31, i, j, k)]) / d->sigma;
          source[d->id(7, i, j, k)] = aux[d->id(34, i, j, k)] * cons[d->id(14, i, j, k)] +
                                      (aux[d->id(33, i, j, k)] * cons[d->id(10, i, j, k)] -
                                      aux[d->id(31, i, j, k)] * cons[d->id(12, i, j, k)]) -
                                      (aux[d->id(23, i, j, k)] - aux[d->id(29, i, j, k)] *
                                      aux[d->id(32, i, j, k)]) / d->sigma;
          source[d->id(8, i, j, k)] = aux[d->id(34, i, j, k)] * cons[d->id(15, i, j, k)] +
                                      (aux[d->id(31, i, j, k)] * cons[d->id(11, i, j, k)] -
                                      aux[d->id(32, i, j, k)] * cons[d->id(10, i, j, k)]) -
                                      (aux[d->id(24, i, j, k)] - aux[d->id(29, i, j, k)] *
                                      aux[d->id(33, i, j, k)]) / d->sigma;
          source[d->id(9, i, j, k)] = aux[d->id(31, i, j, k)] * cons[d->id(13, i, j, k)] +
                                      aux[d->id(32, i, j, k)] * cons[d->id(14, i, j, k)] +
                                      aux[d->id(33, i, j, k)] * cons[d->id(15, i, j, k)] -
                                      (aux[d->id(30, i, j, k)] - aux[d->id(29, i, j, k)] *
                                      aux[d->id(34, i, j, k)]) / d->sigma;
          source[d->id(10, i, j, k)] = 0;
          source[d->id(11, i, j, k)] = 0;
          source[d->id(12, i, j, k)] = 0;
          source[d->id(13, i, j, k)] = - aux[d->id(22, i, j, k)];
          source[d->id(14, i, j, k)] = - aux[d->id(23, i, j, k)];
          source[d->id(15, i, j, k)] = - aux[d->id(24, i, j, k)];
          source[d->id(16, i, j, k)] = aux[d->id(30, i, j, k)] - cons[d->id(16, i, j, k)] / (d->cp * d->cp);
          source[d->id(17, i, j, k)] = - cons[d->id(17, i, j, k)] / (d->cp * d->cp);
        }
      }
    }
  }
}

void TwoFluidEMHD::getPrimitiveVars(double *cons, double *prims, double *aux)
{

}

void TwoFluidEMHD::getPrimitiveVarsSingleCell(double *cons, double *prims, double *aux)
{

}

void TwoFluidEMHD::primsToAll(double *cons, double *prims, double *aux)
{

}



//! Residual function to minimize for cons2prims solver
/*!
    Function to minimize. The minimum of this function (where Z is the independant
  varaible) gives us the approximation for the current value of Z for this species
  of fluid.
*/
static double residual(const double Z, const double StildeSqs, const double Ds, const double tauTildes, double gamma)
{
  // Decalre variables
  double vsq, W, rho, h, p, resid;

  vsq = StildeSqs / (Z * Z);

  // Sanity check
  if (vsq >= 1.0 || Z < 0) return 1.0e6;

  // Continue
  W = 1 / sqrt(1 - vsq);
  rho = Ds / W;
  h = Z / (rho * W * W);
  p = (gamma - 1) * (h - rho) / gamma;

  // Second sanity check
  if (rho < 0 || p < 0 || W < 1 || h < 1) return 1.0e6;

  // Values are physical, compute residual
  resid = (1 - (gamma - 1) / (W * W * gamma)) * Z + ((gamma - 1) / \
          (W * gamma) - 1) * Ds - tauTildes;

  return resid;

}

//! Newton method to solve the (above) residual function
/*!
    Values for StildeSq, D and tauTilde for this species do not vary, hence are
  constant (gamma is also constant but declared a double not const double and I
  dont want to back track through all the code to make consistent---maybe later...)
  Pointer to Z initially holds the guess but this is then modified until it holds
  the solution.
*/
static void newton(double *Z, const double StildeSqs, const double Ds, const double tauTildes, double gamma)
{
  // Rootfind data
  double bestX;
  double x0(*Z);
  double eps(1.0e-4);
  double x1(x0 + eps);
  double tol(1.48e-15);
  double x2;
  double bestF;
  double f0(residual(x0, StildeSqs, Ds, tauTildes, gamma));
  double f1(residual(x1, StildeSqs, Ds, tauTildes, gamma));
  int iter;
  int maxiter(50);
  int found(0);
  // If root can not be found return the best so far
  bestX = x0; bestF = f0;
  for (iter=0; iter<maxiter; iter++) {
    if (fabs(f0) < tol) {
      *Z = x0;
      found = 1;
      break;
    }

    x2 = x1 - f1 * (x1 - x0) / (f1 - f0);
    x1 = x0;
    x0 = x2;
    f1 = f0;
    f0 = residual(x0, StildeSqs, Ds, tauTildes, gamma);
    if (f0 < bestF) {
      bestX = x0;
      bestF = f0;
    }
  }
  if (!found) {
    // Store result of Z=rho*h*W**2
    *Z = bestX;
    printf("Could not find C2P root in %d iterations. Returning %18.16f with residual %18.16f\n", iter, bestX, residual(*Z, StildeSqs, Ds, tauTildes, gamma));
  }
}
