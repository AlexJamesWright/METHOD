#include "hip/hip_runtime.h"
#include "srmhd.h"
#include "weno.h"
#include "cminpack.h"
#include <cmath>
#include <cstdlib>
#include <stdio.h>



SRMHD::SRMHD() : Model()
{
  this->Ncons = 9;
  this->Nprims = 8;
  this->Naux = 10;
}

SRMHD::SRMHD(Data * data) : Model(data)
{
  this->Ncons = (this->data)->Ncons = 9;
  this->Nprims = (this->data)->Nprims = 8;
  this->Naux = (this->data)->Naux = 10;
}


//! Generates the net numerical flux given the current state
/*!
    We are using the flux vector splitting method described in Shu, `Essentially
  Non-Oscillatory and Weighted Essentially Non-Oscillatory Schemes for Hyperbolic
  Conservation Laws`. For the form of the fluxes see Relativistic Magneto..., Anton '10
  with the inclusion of divergence cleaning from Advanced numerical methods for Neutron star
  interfaces, John Muddle.
    Note: We are assuming that all primitive and auxilliary variables are up-to-date
  at the time of this function execution.
*/
void SRMHD::fluxFunc(double *cons, double *prims, double *aux, double *f, double *fnet, int dir)
{
  // Syntax
  Data * d(this->data);

  // up and downwind fluxes
  double *fplus, *fminus;
  hipHostAlloc((void **)&fplus, sizeof(double)*d->Nx*d->Ny*d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&fminus, sizeof(double)*d->Nx*d->Ny*d->Ncons,
                hipHostMallocPortable);

  // Wave speed
  double alpha;
  if (dir == 0) alpha = d->alphaX;
  else alpha = d->alphaY;


  // Order of weno scheme
  int order(2);

  // Generate flux vector
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {

      // Fx: flux in x-direction
      if (dir == 0) {
        // D
        f[d->id(0, i, j)] = cons[d->id(0, i, j)] * prims[d->id(1, i, j)];

        // Sx
        f[d->id(1, i, j)] = cons[d->id(1, i, j)] * prims[d->id(1, i, j)] +
                               prims[d->id(4, i, j)] + aux[d->id(8, i, j)] / 2.0 -
                               aux[d->id(5, i, j)] * prims[d->id(5, i, j)] /
                               aux[d->id(1, i, j)];
        // Sy
        f[d->id(2, i, j)] = cons[d->id(2, i, j)] * prims[d->id(1, i, j)] -
                               aux[d->id(6, i, j)] * prims[d->id(5, i, j)] /
                               aux[d->id(1, i, j)];
        // Sz
        f[d->id(3, i, j)] = cons[d->id(3, i, j)] * prims[d->id(1, i, j)] -
                               aux[d->id(7, i, j)] * prims[d->id(5, i, j)] /
                               aux[d->id(1, i, j)];
        // tau
        f[d->id(4, i, j)] = (cons[d->id(4, i, j)] + prims[d->id(4, i, j)] +
                               aux[d->id(8, i, j)] / 2.0) * prims[d->id(1, i, j)] -
                               aux[d->id(4, i, j)] * prims[d->id(5, i, j)] /
                               aux[d->id(1, i, j)];
        // Bx
        f[d->id(5, i, j)] = cons[d->id(8, i, j)];

        // By
        f[d->id(6, i, j)] = prims[d->id(6, i, j)] * prims[d->id(1, i, j)] -
                               prims[d->id(5, i, j)] * prims[d->id(2, i, j)];
        // Bz
        f[d->id(7, i, j)] = prims[d->id(7, i, j)] * prims[d->id(1, i, j)] -
                               prims[d->id(5, i, j)] * prims[d->id(3, i, j)];
        // Phi
        f[d->id(8, i, j)] = prims[d->id(5, i, j)];

      }

      // Fy: flux in y-direction
      if (dir == 1) {
        // D
        f[d->id(0, i, j)] = cons[d->id(0, i, j)] * prims[d->id(2, i, j)];

        // Sx
        f[d->id(1, i, j)] = cons[d->id(1, i, j)] * prims[d->id(2, i, j)] -
                            aux[d->id(5, i, j)] * prims[d->id(6, i, j)] /
                            aux[d->id(1, i, j)];
        // Sy
        f[d->id(2, i, j)] = cons[d->id(2, i, j)] * prims[d->id(2, i, j)] +
                            prims[d->id(4, i, j)] + aux[d->id(9, i, j)] / 2.0 -
                            aux[d->id(6, i, j)] * prims[d->id(6, i, j)] /
                            aux[d->id(1, i, j)];
        // Sz
        f[d->id(3, i, j)] = cons[d->id(3, i, j)] * prims[d->id(2, i, j)] -
                            aux[d->id(7, i, j)] * prims[d->id(6, i, j)] /
                            aux[d->id(1, i, j)];
        // tau
        f[d->id(4, i, j)] = (cons[d->id(4, i, j)] + prims[d->id(4, i, j)] +
                            aux[d->id(8, i, j)]) * prims[d->id(2, i, j)] -
                            aux[d->id(4, i, j)] * prims[d->id(6, i, j)] /
                            aux[d->id(1, i, j)];
        // Bx
        f[d->id(5, i, j)] = prims[d->id(5, i, j)] * prims[d->id(2, i, j)] -
                            prims[d->id(6, i, j)] * prims[d->id(1, i, j)];
        // By
        f[d->id(6, i, j)] = cons[d->id(8, i, j)];

        // Bz
        f[d->id(7, i, j)] = prims[d->id(7, i, j)] * prims[d->id(2, i, j)] -
                            prims[d->id(6, i, j)] * prims[d->id(3, i, j)];
        // Phi
        f[d->id(8, i, j)] = prims[d->id(6, i, j)];

      }

    } // End j loop
  } // End i loop

  // Lax-Friedrichs approximation of flux
  for (int var(0); var < d->Ncons; var++) {
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        fplus[d->id(var, i, j)] = 0.5 * (f[d->id(var, i, j)] + alpha * cons[d->id(var, i, j)]);
        fminus[d->id(var, i, j)] = 0.5 * (f[d->id(var, i, j)] - alpha * cons[d->id(var, i, j)]);
      }
    }
  }

  // Reconstruct to determine the flux at the cell face and compute difference
  if (dir == 0) { // x-dorection
    for (int var(0); var < d->Ncons; var++) {
      for (int j(0); j < d->Ny; j++) {
        for (int i(order); i < d->Nx-order; i++) {
          fnet[d->id(var, i, j)] = weno3_upwind(fplus[d->id(var, i-order, j)],
                                                fplus[d->id(var, i-order+1, j)],
                                                fplus[d->id(var, i-order+2, j)]) +
                                   weno3_upwind(fminus[d->id(var, i+order-1, j)],
                                                fminus[d->id(var, i+order-2, j)],
                                                fminus[d->id(var, i+order-3, j)]);
        }
      }
    }
  }
  else { // y-direction
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx; i++) {
        for (int j(order); j < d->Ny-order; j++) {
          fnet[d->id(var, i, j)] = weno3_upwind(fplus[d->id(var, i, j-order)],
                                                fplus[d->id(var, i, j-order+1)],
                                                fplus[d->id(var, i, j-order+2)]) +
                                   weno3_upwind(fminus[d->id(var, i, j+order-1)],
                                                fminus[d->id(var, i, j+order-2)],
                                                fminus[d->id(var, i, j+order-3)]);
        }
      }
    }
  }

  // Free arrays
  hipHostFree(fplus);
  hipHostFree(fminus);

}


//! Source required for divergence cleaning
/*!
    See Anton 2010, `Relativistic Magnetohydrodynamcis: Renormalized Eignevectors
  and Full Wave Decompostiion Riemann Solver`
*/
void SRMHD::sourceTerm(double *cons, double *prims, double *aux, double *source)
{
  for (int i(0); i < this->data->Nx; i++) {
    for (int j(0); j < this->data->Ny; j++) {
      for (int var(0); var < this->data->Ncons; var++) {
        if (var == 8) {
          // phi
          source[this->data->id(var, i, j)] = -cons[this->data->id(8, i, j)] / (this->data->cp*this->data->cp);
        }
        else {
          source[this->data->id(var, i, j)] = 0;
        }
      }
    }
  }
}

int residual(void *p, int n, const double *x, double *fvec, int iflag)
{
  // Retrieve additional arguments
  Args * args = (Args*) p;

  // Values must make sense
  if (x[0] >= 1.0 || x[1] < 0) fvec[0] = fvec[1] = 1e6;

  double Bsq(args->Bx*args->Bx + args->By*args->By + args->Bz*args->Bz);
  double Ssq(args->Sx*args->Sx + args->Sy*args->Sy + args->Sz*args->Sz);
  double BS(args->Bx*args->Sx + args->By*args->Sy + args->Bz*args->Sz);
  double W(1 / sqrt(1 - x[0]));
  double rho(args->D / W);
  double h(x[1] / (rho * W * W));
  double pr((h - 1) * rho * (args->g - 1) / args->g);
  if (pr < 0 || rho < 0 || h < 0 || W < 1) fvec[0] = fvec[1] = 1e6;

  // Values should be OK
  fvec[0] = (x[1] + Bsq) * (x[1] + Bsq) * x[0] - (2 * x[1] + Bsq) * BS * BS / (x[1] * x[1]) - Ssq;
  fvec[1] = x[1] + Bsq - pr - Bsq / (2 * W * W) - BS * BS / (2 * x[1] * x[1]) - args->D - args->tau;

  return 0;
}

//! Solve for the primitive and auxilliary variables
/*!
    Method outlined in Anton 2010, `Relativistic Magnetohydrodynamcis:
  Renormalized Eignevectors and Full Wave Decompostiion Riemann Solver`. Requires
  an N=2 rootfind using cminpack library.

  Initial inputs will be the current values of the conserved vector and the
  OLD values for the prims and aux vectors.
  Output will be the current values of cons, prims and aux.
*/
void SRMHD::getPrimitiveVars(double *cons, double *prims, double *aux)
{
  Args args;                          // Additional arguments structure
  const int n(2);                     // Size of system
  double sol[2];                      // Guess and solution vector
  double res[2];                      // Residual/fvec vector
  int info;                           // Rootfinder flag
  // const double tol = 1.49011612e-8;   // Tolerance of rootfinder
  const double tol = 1.49011612e-1;   // Tolerance of rootfinder
  const int lwa = 19;                 // Length of work array = n * (3*n + 13) / 2
  double wa[lwa];                     // Work array

  for (int i(0); i < this->data->Nx; i++) {
    for (int j(0); j < this->data->Ny; j++) {
      info = 9;
      // Set additional args for rootfind
      args.D = cons[this->data->id(0, i, j)];
      args.g = this->data->gamma;
      args.Bx = cons[this->data->id(5, i, j)];
      args.By = cons[this->data->id(6, i, j)];
      args.Bz = cons[this->data->id(7, i, j)];
      args.Sx = cons[this->data->id(1, i, j)];
      args.Sy = cons[this->data->id(2, i, j)];
      args.Sz = cons[this->data->id(3, i, j)];
      args.tau = cons[this->data->id(4, i, j)];

      sol[0] = prims[this->data->id(1, i, j)] * prims[this->data->id(1, i, j)] +
               prims[this->data->id(2, i, j)] * prims[this->data->id(2, i, j)] +
               prims[this->data->id(3, i, j)] * prims[this->data->id(3, i, j)];
      sol[1] = prims[this->data->id(0, i, j)] * aux[this->data->id(0, i, j)] /
               (1 - sol[0]);

      info = __cminpack_func__(hybrd1) (&residual, &args, n, sol, res,
                                        tol, wa, lwa);

      printf("info(%d, %d) = %d\n", i, j, info);

    }
  }

}





//! Generate to the conserved and auxilliary variables
/*!
    Relations have been taken from Anton 2010, `Relativistic Magnetohydrodynamcis:
  Renormalized Eignevectors and Full Wave Decompostiion Riemann Solver`
*/
void SRMHD::primsToAll(double *cons, double *prims, double *aux)
{


  // Syntax
  Data * d = this->data;

  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      // Bx, By, Bz
      d->cons[d->id(5, i, j)] = d->prims[d->id(5, i, j)];
      d->cons[d->id(6, i, j)] = d->prims[d->id(6, i, j)];
      d->cons[d->id(7, i, j)] = d->prims[d->id(7, i, j)];

      // phi
      d->cons[d->id(8, i, j)] = 0;

      // vsq
      d->aux[d->id(9, i, j)] = d->prims[d->id(1, i, j)] * d->prims[d->id(1, i, j)] +
                               d->prims[d->id(2, i, j)] * d->prims[d->id(2, i, j)] +
                                d->prims[d->id(3, i, j)] * d->prims[d->id(3, i, j)];
      // W
      d->aux[d->id(1, i, j)] = 1.0 / sqrt(1 - d->aux[d->id(9, i, j)]);

      // b0
      d->aux[d->id(4, i, j)] = d->aux[d->id(1, i, j)] * (
                               d->prims[d->id(1, i, j)] * d->prims[d->id(5, i, j)] +
                               d->prims[d->id(2, i, j)] * d->prims[d->id(6, i, j)] +
                               d->prims[d->id(3, i, j)] * d->prims[d->id(7, i, j)]);

      // bx, by, bz
      d->aux[d->id(5, i, j)] = d->prims[d->id(5, i, j)] / d->aux[d->id(1, i, j)] +
                               d->aux[d->id(4, i, j)] * d->prims[d->id(1, i, j)];
      d->aux[d->id(6, i, j)] = d->prims[d->id(6, i, j)] / d->aux[d->id(1, i, j)] +
                               d->aux[d->id(4, i, j)] * d->prims[d->id(2, i, j)];
      d->aux[d->id(7, i, j)] = d->prims[d->id(7, i, j)] / d->aux[d->id(1, i, j)] +
                               d->aux[d->id(4, i, j)] * d->prims[d->id(3, i, j)];

      // bsq
      d->aux[d->id(8, i, j)] = (d->prims[d->id(5, i, j)] * d->prims[d->id(5, i, j)] +
                                d->prims[d->id(6, i, j)] * d->prims[d->id(6, i, j)] +
                                d->prims[d->id(7, i, j)] * d->prims[d->id(7, i, j)] +
                                d->aux[d->id(4, i, j)] * d->aux[d->id(4, i, j)]) /
                                (d->aux[d->id(1, i, j)] * d->aux[d->id(1, i, j)]);

      // h
      d->aux[d->id(0, i, j)] = 1 + d->prims[d->id(4, i, j)] / d->prims[d->id(0, i, j)] *
                               (d->gamma / (d->gamma - 1));

      // e
      d->aux[d->id(2, i, j)] = d->prims[d->id(4, i, j)] / (d->prims[d->id(0, i, j)] * (d->gamma - 1));

      // c
      d->aux[d->id(3, i, j)] = sqrt(d->aux[d->id(2, i, j)] * d->gamma * (d->gamma - 1) / d->aux[d->id(0, i, j)]);

      // D
      d->cons[d->id(0, i, j)] = d->prims[d->id(0, i, j)] * d->aux[d->id(1, i, j)];

      // Sx, Sy, Sz
      d->cons[d->id(1, i, j)] = (d->prims[d->id(0, i, j)] * d->aux[d->id(0, i, j)] +
                                 d->aux[d->id(8, i, j)]) * d->aux[d->id(1, i, j)] *
                                 d->aux[d->id(1, i, j)] * d->prims[d->id(1, i, j)] -
                                 d->aux[d->id(4, i, j)] * d->aux[d->id(5, i, j)];
      d->cons[d->id(2, i, j)] = (d->prims[d->id(0, i, j)] * d->aux[d->id(0, i, j)] +
                                 d->aux[d->id(8, i, j)]) * d->aux[d->id(1, i, j)] *
                                 d->aux[d->id(1, i, j)] * d->prims[d->id(2, i, j)] -
                                 d->aux[d->id(4, i, j)] * d->aux[d->id(6, i, j)];
      d->cons[d->id(3, i, j)] = (d->prims[d->id(0, i, j)] * d->aux[d->id(0, i, j)] +
                                 d->aux[d->id(8, i, j)]) * d->aux[d->id(1, i, j)] *
                                 d->aux[d->id(1, i, j)] * d->prims[d->id(3, i, j)] -
                                 d->aux[d->id(4, i, j)] * d->aux[d->id(7, i, j)];
      // tau
      d->cons[d->id(4, i, j)] = (d->prims[d->id(0, i, j)] * d->aux[d->id(0, i, j)] +
                                 d->aux[d->id(8, i, j)]) * d->aux[d->id(1, i, j)] *
                                 d->aux[d->id(1, i, j)] - (d->prims[d->id(4, i, j)] +
                                 d->aux[d->id(8, i, j)] / 2.0) - d->aux[d->id(4, i, j)] *
                                 d->aux[d->id(4, i, j)] - d->cons[d->id(0, i, j)];
      // Alpha (lazy)
      d->alphaX = d->alphaY = 1.0;

    }
  }


}
