#include "hip/hip_runtime.h"
#include "SSP2.h"
#include "srmhd.h"
#include "srrmhd.h"
#include "twoFluidEMHD.h"
#include "deviceArguments.h"
#include "cudaErrorCheck.h"
#include <iostream>
#include <stdexcept>
#include <cstdio>

// Macro for getting array index
#define ID(variable, idx, jdx, kdx) ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx)*(d->Ny)*(d->Nz) + (jdx)*(d->Nz) + (kdx))
#define IDCons(var, idx, jdx, kdx) ( (var) + (idx)*(d->Ncons)*(d->Nz)*(d->Ny) + (jdx)*(d->Ncons)*(d->Nz) + (kdx)*(d->Ncons)  )
#define IDPrims(var, idx, jdx, kdx) ( (var) + (idx)*(d->Nprims)*(d->Nz)*(d->Ny) + (jdx)*(d->Nprims)*(d->Nz) + (kdx)*(d->Nprims)  )
#define IDAux(var, idx, jdx, kdx) ( (var) + (idx)*(d->Naux)*(d->Nz)*(d->Ny) + (jdx)*(d->Naux)*(d->Nz) + (kdx)*(d->Naux)  )

// Device function for stage one of IMEX rootfind
__global__
void stageOne(double * sol, double * cons, double * prims, double * aux, double * source,
              double * wa, double dt, double gam, double tol, int stream,
              int origWidth, int streamWidth, int Ncons, int Nprims, int Naux, int lwa,
              double gamma, double sigma, double mu1, double mu2, double cp,
              ModelType modType_t);

// Device function for stage two of IMEX rootfind
__global__
void stageTwo(double * sol, double * cons, double * prims, double * aux, double * source,
              double * cons1, double * source1, double * flux1,
              double * wa, double dt, double gam, double tol, int stream,
              int origWidth, int streamWidth, int Ncons, int Nprims, int Naux, int lwa,
              double gamma, double sigma, double mu1, double mu2, double cp,
              ModelType modType_t);

//! Residual functions for IMEX SSP2
int IMEX2Residual1(void *p, int n, const double *x, double *fvec, int iflag);
int IMEX2Residual2a(void *p, int n, const double *x, double *fvec, int iflag);
int IMEX2Residual2b(void *p, int n, const double *x, double *fvec, int iflag);

//! Device residual functions for stage one of IMEX SSP2
__device__
int IMEX2Residual1Parallel(void *p, int n, const double *x, double *fvec, int iflag);
//! Device residual functions for stage two A of IMEX SSP2
__device__
int IMEX2Residual2aParallel(void *p, int n, const double *x, double *fvec, int iflag);
//! Device residual functions for stage two of IMEX SSP2
__device__
int IMEX2Residual2Parallel(void *p, int n, const double *x, double *fvec, int iflag);

//! BackwardsRK parameterized constructor
SSP2::SSP2(Data * data, Model * model, Bcs * bc, FluxMethod * fluxMethod) :
              TimeIntegrator(data, model, bc, fluxMethod)

{
  Data * d(this->data);

  this->args = IMEX2Arguments(data);

  lwa = args.lwa;
  Ntot = data->Nx * data->Ny * data->Nz;
  // Hybrd1 variables
  tol = 0.0000000149011612;


  // We only need to implement the integrator on the physical cells provided
  // we apply the boundary conditions to each stage.
  // Determine start and end points
  is = d->Ng;          // i start and end points
  ie = d->Nx - d->Ng;
  if (d->Ny > 1) {
    js = d->Ng;
    je = d->Ny - d->Ng;
  }
  else {
    js = 0;
    je = 1;
  }
  if (d->Nz > 1) {
    ks = d->Ng;
    ke = d->Nz - d->Ng;
  }
  else {
    ks = 0;
    ke = 1;
  }


  // Need work arrays
  hipHostAlloc((void **)&x, sizeof(double) * d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&fvec, sizeof(double) * d->Ncons,
                hipHostMallocPortable);
  hipHostAlloc((void **)&wa, sizeof(double) * lwa,
                hipHostMallocPortable);
  // Interstage results
  hipHostAlloc((void **)&U1, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&U2, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&source1, sizeof(double) * d->Ncons * Ntot,
            hipHostMallocPortable);
  hipHostAlloc((void **)&flux1, sizeof(double) * d->Ncons * Ntot,
            hipHostMallocPortable);
  hipHostAlloc((void **)&source2, sizeof(double) * d->Ncons * Ntot,
            hipHostMallocPortable);
  hipHostAlloc((void **)&flux2, sizeof(double) * d->Ncons * Ntot,
            hipHostMallocPortable);

  // REMOVE WHEN DONE (and in header and destructor)
  hipHostAlloc((void **)&tempCons, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&tempPrims, sizeof(double) * d->Nprims * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&tempAux, sizeof(double) * d->Naux * Ntot,
                  hipHostMallocPortable);
  hipHostAlloc((void **)&tempCons1, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&tempSource1, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&tempFlux1, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&tempSource2, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
  hipHostAlloc((void **)&tempFlux2, sizeof(double) * d->Ncons * Ntot,
                hipHostMallocPortable);
}

SSP2::~SSP2()
{

  // Clean up your mess
  hipHostFree(x);
  hipHostFree(fvec);
  hipHostFree(wa);
  hipHostFree(U1);
  hipHostFree(U2);
  hipHostFree(source1);
  hipHostFree(flux1);
  hipHostFree(source2);
  hipHostFree(flux2);

  hipHostFree(tempCons);
  hipHostFree(tempPrims);
  hipHostFree(tempAux);
  hipHostFree(tempCons1);
  hipHostFree(tempSource1);
  hipHostFree(tempFlux1);
  hipHostFree(tempSource2);
  hipHostFree(tempFlux2);

}

//! Single step functions
void SSP2::step(double * cons, double * prims, double * aux, double dt)
{

  // Syntax
  Data * d(this->data);

  // Get timestep
  if (dt <= 0) dt = d->dt;
  args.dt = dt;


  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        for (int var(0); var < d->Ncons ; var++) tempCons1[ID(var, i, j, k)]  = cons[ID(var, i, j, k)];
        for (int var(0); var < d->Nprims; var++) tempPrims[ID(var, i, j, k)] = prims[ID(var, i, j, k)];
        for (int var(0); var < d->Naux  ; var++) tempAux[ID(var, i, j, k)]   = aux[ID(var, i, j, k)];
      }
    }
  }
  callStageOne(tempCons1, tempPrims, tempAux, tempSource1, dt);
  this->fluxMethod->F(tempCons1, tempPrims, tempAux, d->f, tempFlux1);


  // @todo ###################################### REMEMBER to remove all serial arrays from args when working correctly

  //######## SERIAL CODE #########//
  // Copy data and determine first stage
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        for (int var(0); var < d->Ncons ; var++) x[var]          = cons[ID(var, i, j, k)];
        for (int var(0); var < d->Ncons ; var++) args.cons[var]  = cons[ID(var, i, j, k)];
        for (int var(0); var < d->Nprims; var++) args.prims[var] = prims[ID(var, i, j, k)];
        for (int var(0); var < d->Naux  ; var++) args.aux[var]   = aux[ID(var, i, j, k)];

        args.i = i;
        args.j = j;
        args.k = k;
        // Call hybrd1
        try {
          if ((info = __cminpack_func__(hybrd1)(IMEX2Residual1, this, d->Ncons, x, fvec, tol, wa, lwa)) == 1) {

            for (int var(0); var < d->Ncons; var++)  U1[ID(var, i, j, k)]        = x[var];
          }
          else {
            char s[200];
            sprintf(s, "SSP2 stage 1 failed in cell (%d, %d, %d) with info = %d\nIMEX time integrator could not converge to a solution for stage 1.\n", i, j, k, info);
            throw std::runtime_error(s);
          }
        }
        catch (const std::exception& e) {
          printf("Stage one raises exception with following message:\n%s\n", e.what());
          throw e;
        }
      }
    }
  }

  this->model->getPrimitiveVars(U1, prims, aux);
  this->model->sourceTerm(U1, prims, aux, source1);
  this->fluxMethod->F(U1, prims, aux, d->f, flux1);
  this->bcs->apply(U1);
  this->bcs->apply(flux1);

  printf("Exited stage 1...\n");


  //########################### STAGE TWO #############################//
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        for (int var(0); var < d->Ncons ; var++) tempCons[ID(var, i, j, k)]  = cons[ID(var, i, j, k)];
        for (int var(0); var < d->Nprims; var++) tempPrims[ID(var, i, j, k)] = prims[ID(var, i, j, k)];
        for (int var(0); var < d->Naux  ; var++) tempAux[ID(var, i, j, k)]   = aux[ID(var, i, j, k)];
      }
    }
  }
  printf("Entering stage 2...\n");
  callStageTwo(tempCons, tempPrims, tempAux, tempSource2, tempCons1, tempSource1, tempFlux1, dt);
  this->fluxMethod->F(tempCons, tempPrims, tempAux, d->f, tempFlux2);

  printf("Exited stage2...\n");
  exit(1);

  // Determine solutuion of stage 2
  for (int i(is); i < ie; i++) {
    for (int j(js); j < je; j++) {
      for (int k(ks); k < ke; k++) {
        for (int var(0); var < d->Ncons ; var++) args.cons[var]    = cons[ID(var, i, j, k)];
        for (int var(0); var < d->Nprims; var++) args.prims[var]   = prims[ID(var, i, j, k)];
        for (int var(0); var < d->Naux  ; var++) args.aux[var]     = aux[ID(var, i, j, k)];
        for (int var(0); var < d->Ncons ; var++) args.flux1[var]   = flux1[ID(var,i, j, k)];
        for (int var(0); var < d->Ncons ; var++) args.source1[var] = source1[ID(var, i, j, k)];
        for (int var(0); var < d->Ncons ; var++) x[var]            = U1[ID(var, i, j, k)];
        args.i = i;
        args.j = j;
        args.k = k;

        try {
          // Solve for source terms only
          if ((info = __cminpack_func__(hybrd1)(IMEX2Residual2a, this, d->Ncons, x, fvec, tol, wa, lwa))==1) {
            // Source rootfind successful, euler step flux for stage 2 estimate
            for (int var(0); var < d->Ncons; var++) {
              x[var] = 0.5 * (x[var] + U1[ID(var, i, j, k)] - dt * flux1[ID(var, i, j, k)]);
            }
            try {
              // Solve stage 2
              if ((info = __cminpack_func__(hybrd1)(IMEX2Residual2b, this, d->Ncons, x, fvec, tol, wa, lwa))==1) {
                for (int var(0); var < d->Ncons; var++) U2[ID(var, i, j, k)] = x[var];
              }
              else {
                char s[200];
                sprintf(s, "SSP2 stage 2b failed in cell (%d, %d, %d) with info = %d\nIMEX time integrator could not converge to a solution for stage 2a.\n", i, j, k, info);
                throw std::runtime_error(s);
              }
            }
            catch (const std::exception& e) {
              printf("Stage 2a, U2S, raises exception with following message:\n%s\n", e.what());
              throw e;
            }
          }
          else {
            char s[200];
            sprintf(s, "SSP2 stage 2a failed in cell (%d, %d, %d) with info = %d\nIMEX time integrator could not converge to a solution for stage 2a.\n", i, j, k, info);
            throw std::runtime_error(s);
          }
        }
        catch (const std::exception& e) {
          printf("Stage 2a, U2S, raises exception with following message:\n%s\n", e.what());
          throw e;
        }
      }
    }
  }

  this->bcs->apply(U2, prims, aux);
  this->model->getPrimitiveVars(U2, prims, aux);
  this->model->sourceTerm(U2, prims, aux, source2);
  this->fluxMethod->F(U2, prims, aux, d->f, flux2);
  this->bcs->apply(flux2);


  // Prediction correction
  for (int var(0); var < d->Ncons; var++) {
    for (int i(is); i < ie; i++) {
      for (int j(js); j < je; j++) {
        for (int k(ks); k < ke; k++) {
          cons[ID(var, i, j, k)] = cons[ID(var, i, j, k)] - 0.5 * dt *
                    (flux1[ID(var, i, j, k)] + flux2[ID(var, i, j, k)] -
                    source1[ID(var, i, j, k)] - source2[ID(var, i, j, k)]);
        }
      }
    }
  }
}

void SSP2::callStageOne(double * cons, double * prims, double * aux, double * source, double dt)
{
  Data * d(this->data);
  //########################### STAGE ONE #############################//
  // First need to copy data to the device
  // A single cell requires all cons, prims and aux for the step. Rearrange so
  // we can copy data in contiguous way
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        for (int var(0); var < d->Ncons; var++)  args.cons_h [IDCons(var, i, j, k) ] = cons[ID(var, i, j, k)];
        for (int var(0); var < d->Nprims; var++) args.prims_h[IDPrims(var, i, j, k)] = prims[ID(var, i, j, k)];
        for (int var(0); var < d->Naux; var++)   args.aux_h[IDAux(var, i, j, k)    ] = aux[ID(var, i, j, k)];
      }
    }
  }

  // Data is in correct order, now stream data to the device
  for (int i(0); i < d->Nstreams; i++) {

    // Which cell is at the left bound?
    int lcell(i * d->tpb * d->bpg);
    // Which cell is at the right bound?
    int rcell(lcell + d->tpb * d->bpg);
    if (rcell > d->Ncells) rcell = d->Ncells; // Dont overshoot
    // Memory size to copy in
    int width(rcell - lcell);
    int inMemsize(width * sizeof(double));

    // Send stream's data
    gpuErrchk( hipMemcpyAsync(args.cons_d[i], args.cons_h + lcell*d->Ncons, inMemsize*d->Ncons, hipMemcpyHostToDevice, args.stream[i]) );
    gpuErrchk( hipMemcpyAsync(args.prims_d[i], args.prims_h + lcell*d->Nprims, inMemsize*d->Nprims, hipMemcpyHostToDevice, args.stream[i]) );
    gpuErrchk( hipMemcpyAsync(args.aux_d[i], args.aux_h + lcell*d->Naux, inMemsize*d->Naux, hipMemcpyHostToDevice, args.stream[i]) );

    int sharedMem((d->Ncons + d->Ncons) * sizeof(double) * d->tpb);
    // Call kernel and operate on data
    stageOne <<< d->bpg, d->tpb, sharedMem, args.stream[i] >>>
            (args.sol_d[i], args.cons_d[i], args.prims_d[i], args.aux_d[i],
            args.source_d[i], args.wa_d[i], dt, args.gam, tol, i, d->tpb * d->bpg,
            width, d->Ncons, d->Nprims, d->Naux, lwa,
            d->gamma, d->sigma, d->mu1, d->mu2, d->cp,
            model->modType_t);

    hipStreamSynchronize(args.stream[i]);
    gpuErrchk( hipPeekAtLastError() );

    // Copy all data back
    gpuErrchk( hipMemcpyAsync(args.sol_h + lcell*d->Ncons, args.sol_d[i], inMemsize*d->Ncons, hipMemcpyDeviceToHost, args.stream[i]) );
    gpuErrchk( hipMemcpyAsync(args.prims_h + lcell*d->Nprims, args.prims_d[i], inMemsize*d->Nprims, hipMemcpyDeviceToHost, args.stream[i]) );
    gpuErrchk( hipMemcpyAsync(args.aux_h + lcell*d->Naux, args.aux_d[i], inMemsize*d->Naux, hipMemcpyDeviceToHost, args.stream[i]) );
    gpuErrchk( hipMemcpyAsync(args.source_h + lcell*d->Ncons, args.source_d[i], inMemsize*d->Ncons, hipMemcpyDeviceToHost, args.stream[i]) );
  }
  gpuErrchk( hipDeviceSynchronize() );

  // Rearrange data back into arrays
  for (int i(0); i < d->Nx; i++) {
    for (int j(0); j < d->Ny; j++) {
      for (int k(0); k < d->Nz; k++) {
        for (int var(0); var < d->Ncons; var++)  cons[ID(var, i, j, k)]   = args.sol_h[IDCons(var, i, j, k)];
        for (int var(0); var < d->Ncons; var++)  source[ID(var, i, j, k)] = args.source_h[IDCons(var, i, j, k)];
        for (int var(0); var < d->Nprims; var++) prims[ID(var, i, j, k) ] = args.prims_h[IDPrims(var, i, j, k)];
        for (int var(0); var < d->Naux; var++)   aux[ID(var, i, j, k)]    = args.aux_h[IDAux(var, i, j, k)];
      }
    }
  }
}

__global__
void stageOne(double * sol, double * cons, double * prims, double * aux, double * source,
              double * wa, double dt, double gam, double tol, int stream,
              int origWidth, int streamWidth, int Ncons, int Nprims, int Naux, int lwa,
              double gamma, double sigma, double mu1, double mu2, double cp,
              ModelType modType_t)
{
  const int tID(threadIdx.x);                     //!< thread index (in block)
  const int lID(tID + blockIdx.x * blockDim.x);   //!< local index (in stream)
  const int gID(lID + stream * origWidth);        //!< global index (in domain)
  int info;
  extern __shared__ double sharedArray[];         //!< Shared mem, block specific
  double * fvec  = &sharedArray[tID * 2 * Ncons];
  double * guess = &fvec[Ncons];
  double * WA = &wa[lwa * lID];


  if (lID < streamWidth)
  {
    Model_D * model_d;

    // Store pointers to devuce arrays in the structure
    // to be passed into the residual function
    TimeIntAndModelArgs * args = new TimeIntAndModelArgs(dt, gamma, sigma, mu1, mu2, cp, gam, sol,
                                                         cons, prims, aux, source);
    args->gID = gID;
    // Need to instantiate the correct device model
    switch (modType_t)
    {
      case ModelType::SRMHD:
      //   model = new SRMHD_D();         ################### Need to implement ################
        break;
      case ModelType::SRRMHD:
        model_d = new SRRMHD_D(args);
        break;
      case ModelType::TFEMHD:
      //   model = new twoFluidEMHD_D();  ################### Need to implement ################
        break;
    }


    // First load initial guess (current value of cons)
    for (int i(0); i < Ncons; i++) guess[i] = cons[i + lID * Ncons];
    args->cons = &cons[lID * Ncons];
    args->prims = &prims[lID * Nprims];
    args->aux = &aux[lID * Naux];
    args->source = &source[lID * Ncons];

    // Rootfind
    if ((info = __cminpack_func__(hybrd1)(IMEX2Residual1Parallel, model_d, Ncons, guess, fvec, tol, WA, lwa)) != 1)
    {
      printf("IMEX failed stage 1 for gID %d: info %d\n", gID, info);
    }

    // Copy solution back to sol_d array
    for (int i(0); i < Ncons; i++)
    {
      sol[i] = guess[i];
    }

    // Clean up
    delete args;
    delete model_d;
  }
}

  //! Residual function to minimize for stage one of IMEX SSP2
  /*!
    Root of this function gives the values for U^(1).

    Parameters
    ----------
    p : pointer to BackwardsRK2 object
      The integrator object contains the argument object with the constar, primstar
      etc. arrays and the model object required for the single cell source term
      method.
    n : int
      Size of system
    x : pointer to double
      The array containing the guess
    fvec : pointer to double
      The array containing the residual as a result of the guess x
    iflag : int
      Error flag
  */
  __device__
  int IMEX2Residual1Parallel(void *p, int n, const double *x, double *fvec, int iflag)
  {
    // // Ensure guess is sensible
    // for (int i(0); i < n; i++) {
    //   if (x[i] != x[i])
    //   {
    //     for (int j(0); j<n; j++) fvec[j] = 1e6;
    //     return 0;
    //   }
    // }

    // Cast void pointer
    Model_D * mod = (Model_D *)p;

    // First determine the prim and aux vars due to guess x
    mod->getPrimitiveVarsSingleCell((double *)x, mod->args->prims, mod->args->aux);
    // Determine the source contribution due to the guess x
    mod->sourceTermSingleCell((double *)x, mod->args->prims, mod->args->aux, mod->args->source);

    // Set residual
    for (int i(0); i < n; i++) {
      fvec[i] = x[i] - mod->args->cons[i] - mod->args->dt * mod->args->gam * mod->args->source[i];
      if (mod->args->source[i] != mod->args->source[i] || x[i] != x[i] || fvec[i] != fvec[i])
      {
        for (int j(0); j<n; j++) fvec[j] = 1e6;
        return 0;
      }
    }

    return 0;
  }

  int IMEX2Residual1(void *p, int n, const double *x, double *fvec, int iflag)
  {
    // Cast void pointer
    SSP2 * timeInt = (SSP2 *)p;
    IMEX2Arguments * a(&timeInt->args);

    try {
      // First determine the prim and aux vars due to guess x
      timeInt->model->getPrimitiveVarsSingleCell((double *)x, a->prims, a->aux, a->i, a->j, a->k);
      // Determine the source contribution due to the guess x
      timeInt->model->sourceTermSingleCell((double *)x, a->prims, a->aux, a->source);

      // Set residual
      for (int i(0); i < n; i++) {
        fvec[i] = x[i] - a->cons[i] - a->dt * a->gam * a->source[i];
      }
    }
    catch (const std::exception& e) {
      for (int i(0); i < n; i++) {
        fvec[i] = 1.0e6;
      }
    }

    return 0;
  }


  void SSP2::callStageTwo(double * cons, double * prims, double * aux, double * source, double * cons1, double * source1, double * flux1, double dt)
  {
    Data * d(this->data);
    //########################### STAGE TWO A #############################//
    // First need to copy data to the device
    // A single cell requires all cons, prims and aux for the step. Rearrange so
    // we can copy data in contiguous way
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        for (int k(0); k < d->Nz; k++) {
          for (int var(0); var < d->Ncons; var++)  args.cons_h [IDCons(var, i, j, k)  ] = cons[ID(var, i, j, k)];
          for (int var(0); var < d->Nprims; var++) args.prims_h[IDPrims(var, i, j, k) ] = prims[ID(var, i, j, k)];
          for (int var(0); var < d->Naux; var++)   args.aux_h[IDAux(var, i, j, k)     ] = aux[ID(var, i, j, k)];
          for (int var(0); var < d->Ncons; var++)  args.cons1_h[IDCons(var, i, j, k)  ] = cons1[ID(var, i, j, k)];
          for (int var(0); var < d->Ncons; var++)  args.source1_h[IDCons(var, i, j, k)] = source1[ID(var, i, j, k)];
          for (int var(0); var < d->Ncons; var++)  args.flux1_h[IDCons(var, i, j, k)]   = flux1[ID(var, i, j, k)];
        }
      }
    }

    // Data is in correct order, now stream data to the device
    for (int i(0); i < d->Nstreams; i++) {

      // Which cell is at the left bound?
      int lcell(i * d->tpb * d->bpg);
      // Which cell is at the right bound?
      int rcell(lcell + d->tpb * d->bpg);
      if (rcell > d->Ncells) rcell = d->Ncells; // Dont overshoot
      // Memory size to copy in
      int width(rcell - lcell);
      int inMemsize(width * sizeof(double));

      // Send stream's data
      gpuErrchk( hipMemcpyAsync(args.cons_d[i], args.cons_h + lcell*d->Ncons, inMemsize*d->Ncons, hipMemcpyHostToDevice, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.prims_d[i], args.prims_h + lcell*d->Nprims, inMemsize*d->Nprims, hipMemcpyHostToDevice, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.aux_d[i], args.aux_h + lcell*d->Naux, inMemsize*d->Naux, hipMemcpyHostToDevice, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.cons1_d[i], args.cons1_h + lcell*d->Ncons, inMemsize*d->Ncons, hipMemcpyHostToDevice, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.source1_d[i], args.source1_h + lcell*d->Ncons, inMemsize*d->Ncons, hipMemcpyHostToDevice, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.flux1_d[i], args.flux1_h + lcell*d->Ncons, inMemsize*d->Ncons, hipMemcpyHostToDevice, args.stream[i]) );

      int sharedMem((d->Ncons + d->Ncons) * sizeof(double) * d->tpb);
      // Call kernel and operate on data
      stageTwo <<< d->bpg, d->tpb, sharedMem, args.stream[i] >>>
              (args.sol_d[i], args.cons_d[i], args.prims_d[i], args.aux_d[i], args.source_d[i], args.cons1_d[i],
              args.source1_d[i], args.flux1_d[i], args.wa_d[i], dt, args.gam, tol, i, d->tpb * d->bpg,
              width, d->Ncons, d->Nprims, d->Naux, lwa,
              d->gamma, d->sigma, d->mu1, d->mu2, d->cp,
              model->modType_t);

      hipStreamSynchronize(args.stream[i]);
      gpuErrchk( hipPeekAtLastError() );

      // Copy all data back
      gpuErrchk( hipMemcpyAsync(args.sol_h + lcell*d->Ncons, args.sol_d[i], inMemsize*d->Ncons, hipMemcpyDeviceToHost, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.prims_h + lcell*d->Nprims, args.prims_d[i], inMemsize*d->Nprims, hipMemcpyDeviceToHost, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.aux_h + lcell*d->Naux, args.aux_d[i], inMemsize*d->Naux, hipMemcpyDeviceToHost, args.stream[i]) );
      gpuErrchk( hipMemcpyAsync(args.source_h + lcell*d->Ncons, args.source_d[i], inMemsize*d->Ncons, hipMemcpyDeviceToHost, args.stream[i]) );
    }
    gpuErrchk( hipDeviceSynchronize() );

    // Rearrange data back into arrays
    for (int i(0); i < d->Nx; i++) {
      for (int j(0); j < d->Ny; j++) {
        for (int k(0); k < d->Nz; k++) {
          for (int var(0); var < d->Ncons; var++)  cons[ID(var, i, j, k)]    = args.sol_h[IDCons(var, i, j, k)];
          for (int var(0); var < d->Ncons; var++)  source[ID(var, i, j, k)]  = args.source_h[IDCons(var, i, j, k)];
          for (int var(0); var < d->Nprims; var++) prims[ID(var, i, j, k)  ] = args.prims_h[IDPrims(var, i, j, k)];
          for (int var(0); var < d->Naux; var++)   aux[ID(var, i, j, k)]     = args.aux_h[IDAux(var, i, j, k)];
        }
      }
    }
  }

  __global__
  void stageTwo(double * sol, double * cons, double * prims, double * aux, double * source,
                double * cons1, double * source1, double * flux1,
                double * wa, double dt, double gam, double tol, int stream,
                int origWidth, int streamWidth, int Ncons, int Nprims, int Naux, int lwa,
                double gamma, double sigma, double mu1, double mu2, double cp,
                ModelType modType_t)
{
  const int tID(threadIdx.x);                     //!< thread index (in block)
  const int lID(tID + blockIdx.x * blockDim.x);   //!< local index (in stream)
  const int gID(lID + stream * origWidth);        //!< global index (in domain)
  int info;
  extern __shared__ double sharedArray[];         //!< Shared mem, block specific
  double * fvec  = &sharedArray[tID * 2 * Ncons];
  double * guess = &fvec[Ncons];
  double * WA = &wa[lwa * lID];


  if (lID < streamWidth)
  {
    Model_D * model_d;

    // Store pointers to devuce arrays in the structure
    // to be passed into the residual function
    TimeIntAndModelArgs * args = new TimeIntAndModelArgs(dt, gamma, sigma, mu1, mu2, cp, gam, sol,
                                                         &cons[lID * Ncons], &prims[lID * Nprims],
                                                         &aux[lID * Naux], &source[lID * Ncons],
                                                         &cons1[lID * Ncons], &source1[lID * Ncons],
                                                         &flux1[lID * Ncons]);
    args->gID = gID;
    // Need to instantiate the correct device model
    switch (modType_t)
    {
      case ModelType::SRMHD:
      //   model = new SRMHD_D();         ################### Need to implement ################
        break;
      case ModelType::SRRMHD:
        model_d = new SRRMHD_D(args);
        break;
      case ModelType::TFEMHD:
      //   model = new twoFluidEMHD_D();  ################### Need to implement ################
        break;
    }

    // First load initial guess (current value of cons)
    for (int i(0); i < Ncons; i++) guess[i] = cons[i + lID * Ncons];

    // Rootfind stage 2a
    if ((info = __cminpack_func__(hybrd1)(IMEX2Residual2aParallel, model_d, Ncons, guess, fvec, tol, WA, lwa)) != 1)
    {
      printf("IMEX failed stage 2a for gID %d: info %d\n", gID, info);
    }

    // Construct next guess
    for (int i(0); i < Ncons; i++) guess[i] = 0.5 * (guess[i] + args->cons1[i] - dt*args->flux1[i]);

    // Rootfind stage 2
    if ((info = __cminpack_func__(hybrd1)(IMEX2Residual2Parallel, model_d, Ncons, guess, fvec, tol, WA, lwa)) != 1)
    {
      printf("IMEX failed stage 2 for gID %d: info %d\n", gID, info);
    }

    // Copy solution back to sol_d array
    for (int i(0); i < Ncons; i++)
    {
      sol[i] = guess[i];
    }


    // Clean up
    delete args;
    delete model_d;
  }
}


  //! Residual function to minimize for source contribution in stage two of IMEX SSP2
  /*!
    Root of this function gives the values for Us^(2).

    Parameters
    ----------
    p : pointer to BackwardsRK2 object
      The integrator object contains the argument object with the constar, primstar
      etc. arrays and the model object required for the single cell source term
      method.
    n : int
      Size of system
    x : pointer to double
      The array containing the guess
    fvec : pointer to double
      The array containing the residual as a result of the guess x
    iflag : int
      Error flag
  */
  __device__
int IMEX2Residual2aParallel(void *p, int n, const double *x, double *fvec, int iflag)
{
  // Cast void pointer
  Model_D * mod = (Model_D *)p;

    // First determine the prim and aux vars due to guess x
    mod->getPrimitiveVarsSingleCell((double *)x, mod->args->prims, mod->args->aux);
    // Determine the source contribution due to the guess x
    mod->sourceTermSingleCell((double *)x, mod->args->prims, mod->args->aux, mod->args->source);

    // Set residual
    for (int i(0); i < n; i++)
    {
      fvec[i] = x[i] - mod->args->cons[i] - mod->args->dt * ( (1 - 2*mod->args->gam) * mod->args->source1[i] + mod->args->gam * mod->args->source[i]);
      if (mod->args->source[i] != mod->args->source[i] || x[i] != x[i] || fvec[i] != fvec[i])
      {
        for (int j(0); j<n; j++) fvec[j] = 1e6;
        return 0;
      }
    }

  return 0;
}


__device__
int IMEX2Residual2Parallel(void *p, int n, const double *x, double *fvec, int iflag)
{
// Cast void pointer
Model_D * mod = (Model_D *)p;

  // First determine the prim and aux vars due to guess x
  mod->getPrimitiveVarsSingleCell((double *)x, mod->args->prims, mod->args->aux);
  // Determine the source contribution due to the guess x
  mod->sourceTermSingleCell((double *)x, mod->args->prims, mod->args->aux, mod->args->source);

  // Set residual
  for (int i(0); i < n; i++)
  {
    fvec[i] = x[i] - mod->args->cons[i] + mod->args->dt * (mod->args->flux1[i] - (1 - 2*mod->args->gam) * mod->args->source1[i] - mod->args->gam * mod->args->source[i]);
    if (mod->args->source[i] != mod->args->source[i] || x[i] != x[i] || fvec[i] != fvec[i])
    {
      for (int j(0); j<n; j++) fvec[j] = 1e6;
      return 0;
    }
  }

return 0;
}



  int IMEX2Residual2a(void *p, int n, const double *x, double *fvec, int iflag)
  {
  // Cast void pointer
  SSP2 * timeInt = (SSP2 *)p;
  IMEX2Arguments * a(&timeInt->args);

  try {
    // First determine the prim and aux vars due to guess x
    timeInt->model->getPrimitiveVarsSingleCell((double *)x, a->prims, a->aux, a->i, a->j, a->k);
    // Determine the source contribution due to the guess x
    timeInt->model->sourceTermSingleCell((double *)x, a->prims, a->aux, a->source);

    // Set residual
    for (int i(0); i < n; i++) {
      fvec[i] = x[i] - a->cons[i] - a->dt * ( a->om2gam * a->source1[i] + a->gam * a->source[i]);
    }
  }
  catch (const std::exception& e) {
    for (int i(0); i < n; i++) {
      fvec[i] = 1.0e6;
    }
  }

  return 0;
  }



  //! Residual function to minimize for stage two of IMEX SSP2
  /*!
    Root of this function gives the values for U^(2).

    Parameters
    ----------
    p : pointer to SSP2 object
      The integrator object contains the argument object with the constar, primstar
      etc. arrays and the model object required for the single cell source term
      method.
    n : int
      Size of system
    x : pointer to double
      The array containing the guess
    fvec : pointer to double
      The array containing the residual as a result of the guess x
    iflag : int
      Error flag
  */
  int IMEX2Residual2b(void *p, int n, const double *x, double *fvec, int iflag)
  {
  // Cast void pointer
  SSP2 * timeInt = (SSP2 *)p;
  IMEX2Arguments * a(&timeInt->args);

  try {
    // First determine the prim and aux vars due to guess x
    timeInt->model->getPrimitiveVarsSingleCell((double *)x, a->prims, a->aux, a->i, a->j, a->k);
    // Determine the source contribution due to the guess x
    timeInt->model->sourceTermSingleCell((double *)x, a->prims, a->aux, a->source);
    // Set residual
    for (int i(0); i < n; i++) {
      fvec[i] = x[i] - a->cons[i] + a->dt * (a->flux1[i] - a->om2gam * a->source1[i] - a->gam * a->source[i]);
    }
  }
  catch (const std::exception& e) {
    for (int i(0); i < n; i++) {
      fvec[i] = 1.0e6;
    }
  }


  return 0;
  }
