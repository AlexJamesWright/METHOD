#include "IMEX2Args.h"
#include "cudaErrorCheck.h"

//! Additional arguments parameterized constructor
IMEX2Arguments::IMEX2Arguments(Data * data) : data(data),
                                              gam(0.2928932188134525),
                                              om2gam(0.4142135623730949)
{
  lwa = data->Ncons * (3 * data->Ncons + 13) / 2;

  // Small arrays, no need to malloc
  cons     = new double[data->Ncons ];
  prims    = new double[data->Nprims];
  aux      = new double[data->Naux  ];
  source   = new double[data->Ncons ];
  source1  = new double[data->Ncons ];
  flux1    = new double[data->Ncons ];
  // Alloc host arrays
  gpuErrchk( hipHostAlloc((void **)&cons_h   , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&prims_h  , data->Nprims * data->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&aux_h    , data->Naux   * data->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&source_h , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&cons1_h , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&flux1_h  , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&source1_h, data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&sol_h   , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );

  // Alloc device arrays
  sol_d     = new double*[data->Nstreams];
  cons_d    = new double*[data->Nstreams];
  prims_d   = new double*[data->Nstreams];
  aux_d     = new double*[data->Nstreams];
  source_d  = new double*[data->Nstreams];
  cons1_d  = new double*[data->Nstreams];
  flux1_d   = new double*[data->Nstreams];
  source1_d = new double*[data->Nstreams];
  wa_d      = new double*[data->Nstreams];
  fvec_d      = new double*[data->Nstreams];
  for (int i(0); i < data->Nstreams; i++) {
    gpuErrchk( hipMalloc((void **)&sol_d[i]    , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&cons_d[i]   , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&prims_d[i]  , data->Nprims * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&aux_d[i]    , data->Naux * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&source_d[i] , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&cons1_d[i]  , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&flux1_d[i]  , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&source1_d[i], data->Ncons * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&wa_d[i]     , lwa * data->tpb * data->bpg * sizeof(double)) );
    gpuErrchk( hipMalloc((void **)&fvec_d[i]   , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
  }

  // Create streams
  stream = new hipStream_t[data->Nstreams];
  for (int i(0); i<data->Nstreams; i++) {
    gpuErrchk( hipStreamCreate(&stream[i]) );
  }

  allocd = 1;
}

IMEX2Arguments::~IMEX2Arguments()
{

  delete [] cons;
  delete [] prims;
  delete [] aux;
  delete [] source;
  delete [] source1;
  delete [] flux1;

  for (int i(0); i < data->Nstreams; i++) {
    gpuErrchk( hipFree(sol_d[i]) );
    gpuErrchk( hipFree(cons_d[i]) );
    gpuErrchk( hipFree(prims_d[i]) );
    gpuErrchk( hipFree(aux_d[i]) );
    gpuErrchk( hipFree(source_d[i]) );
    gpuErrchk( hipFree(cons1_d[i]) );
    gpuErrchk( hipFree(flux1_d[i]) );
    gpuErrchk( hipFree(source1_d[i]) );
    gpuErrchk( hipFree(wa_d[i]) );
    gpuErrchk( hipFree(fvec_d[i]) );
  }
  gpuErrchk( hipHostFree(cons_h) );
  gpuErrchk( hipHostFree(prims_h) );
  gpuErrchk( hipHostFree(aux_h) );
  gpuErrchk( hipHostFree(source_h) );
  gpuErrchk( hipHostFree(cons1_h) );
  gpuErrchk( hipHostFree(flux1_h) );
  gpuErrchk( hipHostFree(source1_h) );
  gpuErrchk( hipHostFree(sol_h) );

  allocd = 0;
}

//! Overload assignment operator
IMEX2Arguments& IMEX2Arguments::operator=(const IMEX2Arguments &args)
{
  // Set simulation data
  data = args.data;

  // If no memory has been allocated, allocate
  if (!allocd) {
    lwa = data->Ncons * (3 * data->Ncons + 13) / 2;

    cons     = new double[data->Ncons ];
    prims    = new double[data->Nprims];
    aux      = new double[data->Naux  ];
    source   = new double[data->Ncons ];
    source1  = new double[data->Ncons ];
    flux1    = new double[data->Ncons ];

    // Alloc host arrays
    gpuErrchk( hipHostAlloc((void **)&cons_h   , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&prims_h  , data->Nprims * data->Ncells * sizeof(double), hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&aux_h    , data->Naux   * data->Ncells * sizeof(double), hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&source_h , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&cons1_h  , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&flux1_h  , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&source1_h, data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&sol_h    , data->Ncons  * data->Ncells * sizeof(double), hipHostMallocPortable) );

    // Alloc device arrays
    sol_d     = new double*[data->Nstreams];
    cons_d    = new double*[data->Nstreams];
    prims_d   = new double*[data->Nstreams];
    aux_d     = new double*[data->Nstreams];
    source_d  = new double*[data->Nstreams];
    cons1_d   = new double*[data->Nstreams];
    flux1_d   = new double*[data->Nstreams];
    source1_d = new double*[data->Nstreams];
    wa_d      = new double*[data->Nstreams];
    fvec_d      = new double*[data->Nstreams];
    for (int i(0); i < data->Nstreams; i++) {
      gpuErrchk( hipMalloc((void **)&sol_d[i]    , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&cons_d[i]   , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&prims_d[i]  , data->Nprims * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&aux_d[i]    , data->Naux * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&source_d[i] , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&cons1_d[i]  , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&flux1_d[i]  , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&source1_d[i], data->Ncons * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&wa_d[i]     , lwa * data->tpb * data->bpg * sizeof(double)) );
      gpuErrchk( hipMalloc((void **)&fvec_d[i]   , data->Ncons * data->tpb * data->bpg * sizeof(double)) );
    }

    // Create streams
    stream = new hipStream_t[data->Nstreams];
    for (int i(0); i<data->Nstreams; i++) {
      gpuErrchk( hipStreamCreate(&stream[i]) );
    }

    allocd = 1;

  }

  // Copy accross data
  for (int i(0); i < data->Ncons ; i++) cons [i]   = args.cons [i];
  for (int i(0); i < data->Nprims; i++) prims[i]   = args.prims[i];
  for (int i(0); i < data->Naux  ; i++) aux[i]     = args.aux[i];
  for (int i(0); i < data->Ncons ; i++) source[i]  = args.source[i];
  for (int i(0); i < data->Ncons ; i++) source1[i] = args.source1[i];
  for (int i(0); i < data->Ncons ; i++) flux1[i]   = args.flux1[i];

  return *this;
}
