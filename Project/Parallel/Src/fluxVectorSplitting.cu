#include "hip/hip_runtime.h"
#include "fluxVectorSplitting.h"
#include "cudaErrorCheck.h"
#include <iostream>
#include <cassert>
#include <omp.h>
// Macro for getting array index
#define ID(variable, idx, jdx, kdx)  ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx)*(d->Ny)*(d->Nz) + (jdx)*(d->Nz) + (kdx))
#define IDZ(variable, idx, jdx, kdx) ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx)*(d->Ny)*(d->Nz) + (jdx)*(d->Nz) + (kdx))
#define IDY(variable, idx, jdx, kdx) ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx)*(d->Ny)*(d->Nz) + (jdx) + (kdx)*(d->Ny))
#define IDX(variable, idx, jdx, kdx) ((variable)*(d->Nx)*(d->Ny)*(d->Nz) + (idx) + (jdx)*(d->Nz)*(d->Nx) + (kdx)*(d->Nx))
// Order of WENO scheme
#define ORDER 2



/* NOTES on loading data to device
:
  For loading data into contiguous arrays we need to change the indexing macro.
    In its original form, the data is contiguous in the z-direction, with y and
  then x as the next fastest moving index---so IDZ(var, i, j, k) is identical to
  the normal indexer ID.
    To get data contiguous in y-direction, in a loop assign the elements
  IDY(var, i, j, k) = ID(var, i, j, k), now data is contiguous in y with z then x
  as next fastest moving index.
    Similarly, IDX(var, i, j, k) = ID(var, i, j, k) with arrange data contiguously
  in the x-direction with z and y the next fastest moving index.

  To transform back, apply the same indexing trick.

  Example:
    Flux reconstruction in y direction...

      f = fluxVector(dir = y);

    Rearrange so data is contiguous in y direction...

      for var in Ncons, i in Nx, j in Ny, k in Nz:
        fcontig[IDY(var, i, j, k)] = f[ID(var, i, j, k)]

    Reconstruct flux...

      frecon = fluxRecon(dir = y);

    Copy back data into original form...

      for var in Ncons, i in Nx, j in Ny, k in Nz:
        fnet[ID(var, i, j, k)] = frecon[IDY(var, i, j, k)]
*/





__global__
static void fluxRecon(double * cons, double * f, int stream, int width, double delta, int dir, long unsigned int Ntot)
{

  // Up and downwind fluxes
  extern __shared__ double ftmp [];
  double * fplus = ftmp;
  double * fminus = ftmp + blockDim.x;
  double * frec = ftmp + 2 * blockDim.x;
  const int tID = threadIdx.x;
  const int lID(tID + blockIdx.x * (blockDim.x - 2*ORDER));         // In this stream
  const int gID(lID + stream * (width - 2*ORDER));                  // GlobalID

  // Load data into shared memory and apply Lax-Friedrichs approximation of flux
  if (lID < width) {
    const double tempf = f[lID];
    const double tempc = cons[lID];
    fplus[tID] = 0.5 * (tempf + tempc);
    fminus[tID] = 0.5 * (tempf - tempc);
  }
  __syncthreads();

  if (tID >= ORDER && tID <= blockDim.x-ORDER+1 && lID < width) {
    frec[tID] = weno3_upwind(fplus[tID-ORDER],
                             fplus[tID-ORDER+1],
                             fplus[tID-ORDER+2]) +
                weno3_upwind(fminus[tID+ORDER-1],
                             fminus[tID+ORDER-2],
                             fminus[tID+ORDER-3]);
  }

  //! Now we are going to use the device array 'f' as the differenced, reconstructed flux vector, i.e. fnet in serial code
  __syncthreads();

  if (tID >= ORDER && tID < blockDim.x-ORDER && lID < width && gID < Ntot) {
    f[lID] = frec[tID+1] / delta - frec[tID] / delta;
  }
  __syncthreads();
}



FVS::FVS(Data * data, Model * model, Bcs * bcs) : FluxMethod(data, model, bcs)
{
  // Syntax
  Data * d(this->data);

  // Total number of cells to send
  long unsigned int Ntot(d->Ncons * d->Nx * d->Ny * d->Nz);

  // Define thread set up
  TpB = 512;
  BpG = 128;
  // Resulting size of stream...
  Cwidth = BpG * (TpB - 2*ORDER);
  originalWidth = width = Cwidth + 2*ORDER;

  // ...means we need this many streams
  Nstreams = (Ntot / Cwidth) + 1;

  // Corresponding size of memcpys
  inMemsize = sizeof(double) * width;
  outMemsize = sizeof(double) * Cwidth;

  // Size of dynamically allocd __shared__ memory in device
  sharedMemUsagePerBlock = TpB * 3 * sizeof(double);

  assert(sharedMemUsagePerBlock <= d->prop.sharedMemPerBlock);

  printf("BPG = %d, TPB = %d\n", BpG, TpB);
  printf("Width = %d, Cwidth = %d, Ntot = %lu\n", width, Cwidth, Ntot);
  printf("Shared mem usage = %lu\n", sharedMemUsagePerBlock);

  // Allocate device arrays for each stream
  cons_d = new double*[Nstreams];
  flux_d = new double*[Nstreams];

  for (int i(0); i < Nstreams; i++) {
    gpuErrchk( hipMalloc((void **)&cons_d[i], inMemsize) );
    gpuErrchk( hipMalloc((void **)&flux_d[i], inMemsize) );
  }
  gpuErrchk( hipHostAlloc((void **)&cons_h, Ntot * sizeof(double), hipHostMallocPortable) );
  gpuErrchk( hipHostAlloc((void **)&flux_h, Ntot * sizeof(double), hipHostMallocPortable) );

  // Create streams
  stream = new hipStream_t[Nstreams];
  printf("Created %d streams\n\n\n", Nstreams);
  for (int i(0); i<Nstreams; i++) {
    gpuErrchk( hipStreamCreate(&stream[i]) );
  }

  if (d->Nz > 1) {
    gpuErrchk( hipHostAlloc((void **)&fx, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&fy, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&fz, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable) );
  }
  else if (d->Ny > 1) {
    gpuErrchk( hipHostAlloc((void **)&fx, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable) );
    gpuErrchk( hipHostAlloc((void **)&fy, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable) );
  }
  else {
    gpuErrchk( hipHostAlloc((void **)&fx, sizeof(double) * d->Nx * d->Ny * d->Nz * d->Ncons,
                  hipHostMallocPortable) );
  }
}

FVS::~FVS()
{
  // Syntax
  Data * d(this->data);
  for (int i(0); i < Nstreams; i++) {
    gpuErrchk( hipFree(flux_d[i]) );
    gpuErrchk( hipFree(cons_d[i]) );
  }
  gpuErrchk( hipHostFree(flux_h) );
  gpuErrchk( hipHostFree(cons_h) );
  delete [] flux_d;
  delete [] cons_d;
  delete [] stream;

  if (d->Nz > 1) {
    gpuErrchk( hipHostFree(fx) );
    gpuErrchk( hipHostFree(fy) );
    gpuErrchk( hipHostFree(fz) );
  }
  else if (d->Ny > 1) {
    gpuErrchk( hipHostFree(fx) );
    gpuErrchk( hipHostFree(fy) );
  }
  else {
    gpuErrchk( hipHostFree(fx) );
  }
}


void FVS::fluxReconstruction(double * cons, double * prims, double * aux, double * f, double * frecon, int dir)
{
  // Syntax
  Data * d(this->data);

  double delta;
  // Total number of data points for each vector
  int Ntot(d->Ncons * d->Nx * d->Ny * d->Nz);
  // Get flux vector
  this->model->fluxVector(cons, prims, aux, f, dir);

  // Data must be loaded into device contiguously, so will have to rearrange
  if (dir==0) {
    delta = d->dx;
    for (int var = 0; var<d->Ncons; var++) {
      for (int i = 0; i < d->Nx; i++) {
        for (int j = 0; j < d->Ny; j++) {
          for (int k = 0; k < d->Nz; k++) {
            flux_h[IDX(var, i, j, k)] = f   [ID(var, i, j, k)];
            cons_h[IDX(var, i, j, k)] = cons[ID(var, i, j, k)];
          }
        }
      }
    }
  }
  else if (dir==1) {
    delta = d->dy;
    for (int var = 0; var<d->Ncons; var++) {
      for (int i = 0; i < d->Nx; i++) {
        for (int j = 0; j < d->Ny; j++) {
          for (int k = 0; k < d->Nz; k++) {
            flux_h[IDY(var, i, j, k)] = f   [ID(var, i, j, k)];
            cons_h[IDY(var, i, j, k)] = cons[ID(var, i, j, k)];
          }
        }
      }
    }
  }
  else {
    delta = d->dz;
    for (int var = 0; var<d->Ncons; var++) {
      for (int i = 0; i < d->Nx; i++) {
        for (int j = 0; j < d->Ny; j++) {
          for (int k = 0; k < d->Nz; k++) {
            flux_h[IDZ(var, i, j, k)] = f   [ID(var, i, j, k)];
            cons_h[IDZ(var, i, j, k)] = cons[ID(var, i, j, k)];
          }
        }
      }
    }
  }

  // Data is now contiguous, send to GPU and do work

  int lb, rb; // Left and right boundary of data sent to device
  // Set/Reset width and memsize...
  Cwidth = BpG * (TpB - 2*ORDER);
  width = Cwidth + 2*ORDER;

  // Corresponding size of memcpys
  inMemsize = sizeof(double) * width;
  outMemsize = sizeof(double) * Cwidth;

  // Call parallel reconstruction
  for (int i(0); i<Nstreams; i++) {

    // First determine where in the contiguous array the left boundary of this stream corresponds to
    lb = i*(width - 2 * ORDER);
    rb = lb + width;
    if (i == Nstreams-1) {
      rb = Ntot;
      // Final stream so only do remaining cells
      width = rb - lb;
      Cwidth = width - 2*ORDER;
      inMemsize = sizeof(double) * width;
      outMemsize = sizeof(double) * Cwidth;
    }
    // Copy stream's data to device
    gpuErrchk( hipMemcpyAsync(cons_d[i], cons_h + lb, inMemsize, hipMemcpyHostToDevice, stream[i]) );
    gpuErrchk( hipMemcpyAsync(flux_d[i], flux_h + lb, inMemsize, hipMemcpyHostToDevice, stream[i]) );

    fluxRecon<<<BpG, TpB, sharedMemUsagePerBlock, stream[i]>>>(cons_d[i], flux_d[i], i, originalWidth, delta, dir, Ntot);

    gpuErrchk( hipMemcpyAsync(flux_h+lb+ORDER, flux_d[i]+ORDER, outMemsize, hipMemcpyDeviceToHost, stream[i]) );
    gpuErrchk( hipPeekAtLastError() );
  }
  gpuErrchk( hipDeviceSynchronize() );

    // Data must be loaded back into original order on the host
    if (dir==0) {
      for (int var = 0; var<d->Ncons; var++) {
        for (int i = 0; i < d->Nx; i++) {
          for (int j = 0; j < d->Ny; j++) {
            for (int k = 0; k < d->Nz; k++) {
              frecon[ID(var, i, j, k)] = flux_h[IDX(var, i, j, k)];
            }
          }
        }
      }
    }
    else if (dir==1) {
      for (int var = 0; var<d->Ncons; var++) {
        for (int i = 0; i < d->Nx; i++) {
          for (int j = 0; j < d->Ny; j++) {
            for (int k = 0; k < d->Nz; k++) {
              frecon[ID(var, i, j, k)] = flux_h[IDY(var, i, j, k)];
            }
          }
        }
      }
    }
    else {
      for (int var = 0; var<d->Ncons; var++) {
        for (int i = 0; i < d->Nx; i++) {
          for (int j = 0; j < d->Ny; j++) {
            for (int k = 0; k < d->Nz; k++) {
              frecon[ID(var, i, j, k)] = flux_h[IDZ(var, i, j, k)];
            }
          }
        }
      }
    }
}

void FVS::F(double * cons, double * prims, double * aux, double * f, double * fnet)
{
  // Syntax
  Data * d(this->data);

  // 3D domain, loop over all cells determining the net flux
  if (d->Ny > 1 && d->Nz > 1) {

    // Determine flux vectors
    this->fluxReconstruction(cons, prims, aux, f, fx, 0);
    this->fluxReconstruction(cons, prims, aux, f, fy, 1);
    this->fluxReconstruction(cons, prims, aux, f, fz, 2);
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
        for (int j(0); j < d->Ny-1; j++) {
          for (int k(0); k < d->Nz-1; k++) {
            fnet[ID(var, i, j, k)] = fx[ID(var, i, j, k)] + fy[ID(var, i, j, k)] + fz[ID(var, i, j, k)];
          }
        }
      }
    }
  }

  // 2D domain, loop over x- and y-directions determining the net flux
  else if (d->Ny > 1) {
    this->fluxReconstruction(cons, prims, aux, f, fx, 0);
    this->fluxReconstruction(cons, prims, aux, f, fy, 1);
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
        for (int j(0); j < d->Ny-1; j++) {
          fnet[ID(var, i, j, 0)] = fx[ID(var, i, j, 0)] + fy[ID(var, i, j, 0)];
        }
      }
    }
  }

  // Otherwise, domain is 1D only loop over x direction
  else {
    this->fluxReconstruction(cons, prims, aux, f, fx, 0);
    for (int var(0); var < d->Ncons; var++) {
      for (int i(0); i < d->Nx-1; i++) {
          fnet[ID(var, i, 0, 0)] = fx[ID(var, i, 0, 0)];
      }
    }
  }
}
